#include "hip/hip_runtime.h"
#include "opt-sched/Scheduler/bb_spill.h"
#include "opt-sched/Scheduler/aco.h"
#include "opt-sched/Scheduler/config.h"
#include "opt-sched/Scheduler/data_dep.h"
#include "opt-sched/Scheduler/enumerator.h"
#include "opt-sched/Scheduler/list_sched.h"
#include "opt-sched/Scheduler/logger.h"
#include "opt-sched/Scheduler/random.h"
#include "opt-sched/Scheduler/reg_alloc.h"
#include "opt-sched/Scheduler/register.h"
#include "opt-sched/Scheduler/relaxed_sched.h"
#include "opt-sched/Scheduler/stats.h"
#include "opt-sched/Scheduler/utilities.h"
#include <algorithm>
#include <cstdio>
#include <iostream>
#include <map>
#include <numeric>
#include <set>
#include <sstream>
#include <utility>
#include <hip/hip_runtime.h>

extern bool OPTSCHED_gPrintSpills;

using namespace llvm::opt_sched;

// The denominator used when calculating cost weight.
static const int COST_WGHT_BASE = 10;

BBWithSpill::BBWithSpill(const OptSchedTarget *OST_, DataDepGraph *dataDepGraph,
                         long rgnNum, int16_t sigHashSize, LB_ALG lbAlg,
                         SchedPriorities hurstcPrirts,
                         SchedPriorities enumPrirts, bool vrfySched,
                         Pruning PruningStrategy, bool SchedForRPOnly,
                         bool enblStallEnum, int SCW,
                         SPILL_COST_FUNCTION spillCostFunc,
                         SchedulerType HeurSchedType)
    : SchedRegion(OST_->MM, dataDepGraph, rgnNum, sigHashSize, lbAlg,
                  hurstcPrirts, enumPrirts, vrfySched, PruningStrategy,
                  HeurSchedType, spillCostFunc),
      OST(OST_) {
  enumrtr_ = NULL;
  optmlSpillCost_ = INVALID_VALUE;

  crntCycleNum_ = INVALID_VALUE;
  crntSlotNum_ = INVALID_VALUE;
  crntSpillCost_ = INVALID_VALUE;

  SchedForRPOnly_ = SchedForRPOnly;

  enblStallEnum_ = enblStallEnum;
  SCW_ = SCW;
  schedCostFactor_ = COST_WGHT_BASE;
  trackLiveRangeLngths_ = true;

  regTypeCnt_ = OST->MM->GetRegTypeCnt();
  regFiles_ = dataDepGraph->getRegFiles();
  liveRegs_ = new WeightedBitVector[regTypeCnt_];
  livePhysRegs_ = new WeightedBitVector[regTypeCnt_];
  spillCosts_ = new InstCount[dataDepGraph_->GetInstCnt()];
  peakRegPressures_ = new InstCount[regTypeCnt_];
  regPressures_ = new unsigned[regTypeCnt_];
  sumOfLiveIntervalLengths_size_ = regTypeCnt_;
  sumOfLiveIntervalLengths_ = new int[sumOfLiveIntervalLengths_size_];

  //initialize all values to 0
  for (int i = 0; i < sumOfLiveIntervalLengths_size_; i++)
    sumOfLiveIntervalLengths_[i] = 0;

  entryInstCnt_ = 0;
  exitInstCnt_ = 0;
  schduldEntryInstCnt_ = 0;
  schduldExitInstCnt_ = 0;
  schduldInstCnt_ = 0;
}
/****************************************************************************/

BBWithSpill::~BBWithSpill() {
  if (enumrtr_ != NULL) {
    delete enumrtr_;
  }
 
  delete[] regPressures_;
  delete[] sumOfLiveIntervalLengths_;
  delete[] liveRegs_;
  delete[] livePhysRegs_;
  delete[] spillCosts_;
  delete[] peakRegPressures_;
}
/*****************************************************************************/

bool BBWithSpill::EnableEnum_() {
  return true;
  /*
  if (maxSpillCost_ > 0 && hurstcCost_ > maxSpillCost_) {
    Logger::Info("Bypassing enumeration due to a large spill cost of %d",
                 hurstcCost_);
    return false;
  }
  return true;
  */
}
/*****************************************************************************/

ConstrainedScheduler *BBWithSpill::AllocHeuristicScheduler_() {
  switch (GetHeuristicSchedulerType()) {
  case SCHED_LIST:
    return new ListScheduler(dataDepGraph_, machMdl_, abslutSchedUprBound_,
                             GetHeuristicPriorities());
    break;
  case SCHED_SEQ:
    return new SequentialListScheduler(dataDepGraph_, machMdl_,
                                       abslutSchedUprBound_,
                                       GetHeuristicPriorities());
    break;
  }
  llvm_unreachable("Unknown heuristic scheduler type!");
}
/*****************************************************************************/

void BBWithSpill::SetupPhysRegs_() {
  int physRegCnt;
  for (int i = 0; i < regTypeCnt_; i++) {
    physRegCnt = regFiles_[i].FindPhysRegCnt();
    if (physRegCnt > 0)
      livePhysRegs_[i].Construct(physRegCnt);
  }
}
/*****************************************************************************/

void BBWithSpill::CmputAbslutUprBound_() {
  abslutSchedUprBound_ = dataDepGraph_->GetAbslutSchedUprBound();
  dataDepGraph_->SetAbslutSchedUprBound(abslutSchedUprBound_);
}
/*****************************************************************************/

void BBWithSpill::CmputSchedUprBound_() {
  // The maximum increase in sched length that might result in a smaller cost
  // than the known one
  int maxLngthIncrmnt = (GetBestCost() - 1) / schedCostFactor_;

  if (machMdl_->IsSimple() && dataDepGraph_->GetMaxLtncy() <= 1) {
#if defined(IS_DEBUG_DAG) || defined(IS_DEBUG_SIMPLE_DAGS)
    Logger::Info("Simple DAG with max latency of one or less.");
#endif
    maxLngthIncrmnt = 0;
  }

  assert(maxLngthIncrmnt >= 0);

  // Any schedule longer than this will have a cost that is greater than or
  // equal to that of the list schedule
  schedUprBound_ = schedLwrBound_ + maxLngthIncrmnt;

  if (abslutSchedUprBound_ < schedUprBound_) {
    schedUprBound_ = abslutSchedUprBound_;
  }
}
/*****************************************************************************/

static InstCount ComputeSLILStaticLowerBound(int64_t regTypeCnt_,
                                             RegisterFile *regFiles_,
                                             DataDepGraph *dataDepGraph_) {
  // (Chris): To calculate a naive lower bound of the SLIL, count all the defs
  // and uses for each register.
  int naiveLowerBound = 0;
  for (int i = 0; i < regTypeCnt_; ++i) {
    for (int j = 0; j < regFiles_[i].GetRegCnt(); ++j) {
      const auto &reg = regFiles_[i].GetReg(j);
      for (const auto &instruction : reg->GetDefList()) {
        if (reg->AddToInterval(instruction)) {
          ++naiveLowerBound;
        }
      }
      for (const auto &instruction : reg->GetUseList()) {
        if (reg->AddToInterval(instruction)) {
          ++naiveLowerBound;
        }
      }
    }
  }

#if defined(IS_DEBUG_SLIL_COST_LOWER_BOUND)
  Logger::Info("SLIL Naive Static Lower Bound Cost  is %llu for Dag %s",
               naiveLowerBound, dataDepGraph_->GetDagID());
#endif

  // (Chris): Another improvement to the lower bound calculation takes advantage
  // of the transitive closure of the DAG. Suppose instruction X must happen
  // between A and B, where A defines a register that B uses. Then, the live
  // range length of A increases by 1.
  auto closureLowerBound = naiveLowerBound;
  for (int i = 0; i < dataDepGraph_->GetInstCnt(); ++i) {
    const auto &inst = dataDepGraph_->GetInstByIndx(i);
    // For each register this instruction defines, compute the intersection
    // between the recursive successor list of this instruction and the
    // recursive predecessors of the dependent instruction.
    Register **definedRegisters = nullptr;
    auto defRegCount = inst->GetDefs(definedRegisters);
    auto recSuccBV = inst->GetRcrsvNghbrBitVector(DIR_FRWRD);
    for (int j = 0; j < defRegCount; ++j) {
      for (const auto &dependentInst : definedRegisters[j]->GetUseList()) {
        auto recPredBV = const_cast<SchedInstruction *>(dependentInst)
                             ->GetRcrsvNghbrBitVector(DIR_BKWRD);
        assert(recSuccBV->GetSize() == recPredBV->GetSize() &&
               "Successor list size doesn't match predecessor list size!");
        for (int k = 0; k < recSuccBV->GetSize(); ++k) {
          if (recSuccBV->GetBit(k) & recPredBV->GetBit(k)) {
            if (definedRegisters[j]->AddToInterval(
                    dataDepGraph_->GetInstByIndx(k))) {
              ++closureLowerBound;
            }
          }
        }
      }
    }
  }

#if defined(IS_DEBUG_SLIL_COST_LOWER_BOUND)
  Logger::Info("SLIL Closur Static Lower Bound Cost is %llu for Dag %s",
               closureLowerBound, dataDepGraph_->GetDagID());
#endif

  // (Chris): A better lower bound can be computed by adding more to the SLIL
  // based on the instructions that use more than one register (defined by
  // different instructions).
  int commonUseLowerBound = closureLowerBound;
  std::vector<std::pair<const SchedInstruction *, Register *>> usedInsts;
  for (int i = 0; i < dataDepGraph_->GetInstCnt(); ++i) {
    const auto &inst = dataDepGraph_->GetInstByIndx(i);
    Register **usedRegisters = nullptr;
    auto usedRegCount = inst->GetUses(usedRegisters);

    // Get a list of instructions that define the registers, in array form.
    usedInsts.clear();
    for (int j = 0; j < usedRegCount; ++j) {
      Register *reg = usedRegisters[j];
      assert(reg->GetDefList().size() == 1 &&
             "Number of defs for register is not 1!");
      usedInsts.push_back(std::make_pair(*(reg->GetDefList().begin()), reg));
    }

#if defined(IS_DEBUG_SLIL_COMMON_USE_LB)
    Logger::Info("Common Use Lower Bound Instruction %d", inst->GetNum());
    Logger::Info("  Instruction %d uses:", inst->GetNum());
    for (const auto &p : usedInsts) {
      Logger::Info("    Instruction %d register %d:%d", p.first->GetNum(),
                   p.second->GetType(), p.second->GetNum());
    }

    for (const auto &p : usedInsts) {
      Logger::Info("  Live interval of Register %d:%d (defined by Inst %d):",
                   p.second->GetType(), p.second->GetNum(), p.first->GetNum());
      for (const auto &s : p.second->GetLiveInterval()) {
        Logger::Info("    %d", s->GetNum());
      }
    }
#endif

    for (size_t j = 0; j < usedInsts.size(); ++j) {
      for (size_t k = j + 1; k < usedInsts.size(); ++k) {
        const auto &jReg = usedInsts[j].second;
        const auto &kReg = usedInsts[k].second;

        // If k is not in the live interval of j AND ALSO j is not in the live
        // interval of k, add k to the live interval of j, and increment the
        // lower bound by 1.
        bool found = jReg->IsInInterval(usedInsts[k].first) ||
                     kReg->IsInInterval(usedInsts[j].first) ||
                     jReg->IsInPossibleInterval(usedInsts[k].first) ||
                     kReg->IsInPossibleInterval(usedInsts[j].first);

        if (!found && usedInsts[j].first != usedInsts[k].first) {
          jReg->AddToPossibleInterval(usedInsts[k].first);
          kReg->AddToPossibleInterval(usedInsts[j].first);

          commonUseLowerBound++;
#if defined(IS_DEBUG_SLIL_COMMON_USE_LB)
          Logger::Info("  Common Use: Found two instructions %d and %d",
                       usedInsts[j].first->GetNum(),
                       usedInsts[k].first->GetNum());
#endif
        }
      }
    }
  }

#if defined(IS_DEBUG_SLIL_COST_LOWER_BOUND)
  if (commonUseLowerBound > closureLowerBound)
    Logger::Info("SLIL Final  Static Lower Bound Cost is %llu for Dag %s",
                 commonUseLowerBound, dataDepGraph_->GetDagID());
#endif

  return static_cast<InstCount>(commonUseLowerBound);
}
/*****************************************************************************/

InstCount BBWithSpill::CmputCostLwrBound() {
  InstCount spillCostLwrBound = 0;

  if (GetSpillCostFunc() == SCF_SLIL) {
    spillCostLwrBound =
        ComputeSLILStaticLowerBound(regTypeCnt_, regFiles_, dataDepGraph_);
    dynamicSlilLowerBound_ = spillCostLwrBound;
    staticSlilLowerBound_ = spillCostLwrBound;
  }

  // for(InstCount i=0; i< dataDepGraph_->GetInstCnt(); i++) {
  //   inst = dataDepGraph_->GetInstByIndx(i);
  // }

  InstCount staticLowerBound =
      schedLwrBound_ * schedCostFactor_ + spillCostLwrBound * SCW_;

#if defined(IS_DEBUG_STATIC_LOWER_BOUND)
  Logger::Info(
      "DAG %s spillCostLB %d scFactor %d lengthLB %d lenFactor %d staticLB %d",
      dataDepGraph_->GetDagID(), spillCostLwrBound, SCW_, schedLwrBound_,
      schedCostFactor_, staticLowerBound);
#endif

  return staticLowerBound;
}
/*****************************************************************************/

__host__ __device__
void BBWithSpill::InitForSchdulng() {
  InitForCostCmputtn_();

  schduldEntryInstCnt_ = 0;
  schduldExitInstCnt_ = 0;
  schduldInstCnt_ = 0;
}

//device version of InitForSchdulng(), bypasses needles polymorphism that
//prevents it from being called on device
__device__
void BBWithSpill::Dev_InitForSchdulng() {
  InitForCostCmputtn_();

  schduldEntryInstCnt_ = 0;
  schduldExitInstCnt_ = 0;
  schduldInstCnt_ = 0;
}

/*****************************************************************************/

__host__ __device__
void BBWithSpill::InitForCostCmputtn_() {
  int i;

  crntCycleNum_ = 0;
  crntSlotNum_ = 0;
  crntSpillCost_ = 0;
  crntStepNum_ = -1;
  peakSpillCost_ = 0;
  totSpillCost_ = 0;

  for (i = 0; i < regTypeCnt_; i++) {
    regFiles_[i].ResetCrntUseCnts();
    regFiles_[i].ResetCrntLngths();
  }

  for (i = 0; i < regTypeCnt_; i++) {
#ifdef __CUDA_ARCH__
    liveRegs_[i].Dev_Reset();
#else
    liveRegs_[i].Reset();
#endif

    if (regFiles_[i].GetPhysRegCnt() > 0) {
#ifdef __CUDA_ARCH__
      livePhysRegs_[i].Dev_Reset();
#else
      livePhysRegs_[i].Reset();
#endif
    }

    //    if (chkCnflcts_)
    //      regFiles_[i].ResetConflicts();
    peakRegPressures_[i] = 0;
    regPressures_[i] = 0;
  }

  for (i = 0; i < dataDepGraph_->GetInstCnt(); i++)
    spillCosts_[i] = 0;

  for (int i = 0; i < sumOfLiveIntervalLengths_size_; i++)
    sumOfLiveIntervalLengths_[i] = 0;

  dynamicSlilLowerBound_ = staticSlilLowerBound_;
}
/*****************************************************************************/

InstCount BBWithSpill::CmputNormCost_(InstSchedule *sched,
                                      COST_COMP_MODE compMode,
                                      InstCount &execCost, bool trackCnflcts) {
  InstCount cost = CmputCost_(sched, compMode, execCost, trackCnflcts);

  cost -= GetCostLwrBound();
  execCost -= GetCostLwrBound();

  sched->SetCost(cost);
  sched->SetExecCost(execCost);
  return cost;
}
/*****************************************************************************/

InstCount BBWithSpill::CmputCost_(InstSchedule *sched, COST_COMP_MODE compMode,
                                  InstCount &execCost, bool trackCnflcts) {
  if (compMode == CCM_STTC) {
    if (GetSpillCostFunc() == SCF_SPILLS) {
      LocalRegAlloc regAlloc(sched, dataDepGraph_);
      regAlloc.SetupForRegAlloc();
      regAlloc.AllocRegs();
      crntSpillCost_ = regAlloc.GetCost();
    }
  }

  assert(sched->IsComplete());
  InstCount cost = sched->GetCrntLngth() * schedCostFactor_;
  execCost = cost;
  cost += crntSpillCost_ * SCW_;
  sched->SetSpillCosts(spillCosts_);
  sched->SetPeakRegPressures(peakRegPressures_);
  sched->SetSpillCost(crntSpillCost_);
  return cost;
}
/*****************************************************************************/

__host__ __device__
void BBWithSpill::CmputCrntSpillCost_() {
  switch (GetSpillCostFunc()) {
  case SCF_PERP:
  case SCF_PRP:
  case SCF_PEAK_PER_TYPE:
  case SCF_TARGET:
    crntSpillCost_ = peakSpillCost_;
    break;
  case SCF_SUM:
    crntSpillCost_ = totSpillCost_;
    break;
  case SCF_PEAK_PLUS_AVG:
    crntSpillCost_ =
        peakSpillCost_ + totSpillCost_ / dataDepGraph_->GetInstCnt();
    break;
  case SCF_SLIL:
    crntSpillCost_ = slilSpillCost_;
    break;
  default:
    crntSpillCost_ = peakSpillCost_;
    break;
  }
}
/*****************************************************************************/
//note: Logger::info/fatal cannot be called on device. using __CUDA_ARCH__ 
//macro to call printf on device instead
__host__ __device__
void BBWithSpill::UpdateSpillInfoForSchdul_(SchedInstruction *inst,
                                            bool trackCnflcts) {
  int16_t regType;
  int defCnt, useCnt, regNum, physRegNum;
  Register **defs, **uses;
  Register *def, *use;
  int liveRegs;
  InstCount newSpillCost;
  int accumulator;

  defCnt = inst->GetDefs(defs);
  useCnt = inst->GetUses(uses);

#ifdef IS_DEBUG_REG_PRESSURE
  Logger::Info("Updating reg pressure after scheduling Inst %d",
               inst->GetNum());
#endif

  // Update Live regs after uses
  for (int i = 0; i < useCnt; i++) {
    use = uses[i];
    regType = use->GetType();
    regNum = use->GetNum();
    physRegNum = use->GetPhysicalNumber();

    if (use->IsLive() == false) {
#ifdef __CUDA_ARCH__
      printf("Reg %d of type %d is used without being defined\n", regNum, 
	     regType);
#else
      Logger::Fatal("Reg %d of type %d is used without being defined", regNum,
                    regType);
#endif
    }

#ifdef IS_DEBUG_REG_PRESSURE
#ifdef __CUDA_ARCH__
    printf("Inst %d uses reg %d of type %d and %d uses\n", inst->GetNum(), regNum, regType, use->GetUseCnt());
#else
    Logger::Info("Inst %d uses reg %d of type %d and %d uses", inst->GetNum(),
                 regNum, regType, use->GetUseCnt());
#endif
#endif

    use->AddCrntUse();

    if (use->IsLive() == false) {
      // (Chris): The SLIL calculation below the def and use for-loops doesn't
      // consider the last use of a register. Thus, an additional increment must
      // happen here.
      if (GetSpillCostFunc() == SCF_SLIL) {
        sumOfLiveIntervalLengths_[regType]++;
        if (!use->IsInInterval(inst) && !use->IsInPossibleInterval(inst)) {
          ++dynamicSlilLowerBound_;
        }
      }

      liveRegs_[regType].SetBit(regNum, false, use->GetWght());

#ifdef IS_DEBUG_REG_PRESSURE
#ifdef __CUDA_ARCH__
      printf("Reg type %d now has %d live regs\n", regType, 
	     liveRegs_[regType].GetOneCnt());
#else
      Logger::Info("Reg type %d now has %d live regs", regType,
                   liveRegs_[regType].GetOneCnt());
#endif
#endif

      if (regFiles_[regType].GetPhysRegCnt() > 0 && physRegNum >= 0)
        livePhysRegs_[regType].SetBit(physRegNum, false, use->GetWght());
    }
  }

  // Update Live regs after defs
  for (int i = 0; i < defCnt; i++) {
    def = defs[i];
    regType = def->GetType();
    regNum = def->GetNum();
    physRegNum = def->GetPhysicalNumber();

#ifdef IS_DEBUG_REG_PRESSURE
#ifdef __CUDA_ARCH__
    printf("Inst %d defines reg %d of type %d and %d uses\n",inst->GetNum(), 
	   regNum, regType, def->GetUseCnt());
#else
    Logger::Info("Inst %d defines reg %d of type %d and %d uses",
                 inst->GetNum(), regNum, regType, def->GetUseCnt());
#endif
#endif

    // if (def->GetUseCnt() > 0) {

    if (trackCnflcts && liveRegs_[regType].GetOneCnt() > 0)
      regFiles_[regType].AddConflictsWithLiveRegs(
          regNum, liveRegs_[regType].GetOneCnt());

    liveRegs_[regType].SetBit(regNum, true, def->GetWght());

#ifdef IS_DEBUG_REG_PRESSURE
#ifdef __CUDA_ARCH__
    printf("Reg type %d now has %d live regs\n", regType,
           liveRegs_[regType].GetOneCnt());
#else
    Logger::Info("Reg type %d now has %d live regs", regType,
                 liveRegs_[regType].GetOneCnt());
#endif
#endif

    if (regFiles_[regType].GetPhysRegCnt() > 0 && physRegNum >= 0)
      livePhysRegs_[regType].SetBit(physRegNum, true, def->GetWght());
    def->ResetCrntUseCnt();
    //}
  }

  newSpillCost = 0;

#ifdef IS_DEBUG_SLIL_CORRECT
  if (OPTSCHED_gPrintSpills) {
#ifdef __CUDA_ARCH__
    printf("Printing live range lengths for instruction BEFORE calculation.\n");
#else
    Logger::Info(
        "Printing live range lengths for instruction BEFORE calculation.");
#endif
    for (int j = 0; j < sumOfLiveIntervalLengths_size_; j++) {
#ifdef __CUDA_ARCH__
      printf("SLIL for regType %d %s is currently %d\n", j,
	     sumOfLiveIntervalLengths_[j]);
#else
      Logger::Info("SLIL for regType %d %s is currently %d", j,
                   sumOfLiveIntervalLengths_[j]);
#endif
    }
#ifdef __CUDA_ARCH__
    printf();
#else
    Logger::Info("Now computing spill cost for instruction.");
#endif
  }
#endif

  for (int16_t i = 0; i < regTypeCnt_; i++) {
    liveRegs = liveRegs_[i].GetWghtedCnt();
    // Set current RP for register type "i"
    regPressures_[i] = liveRegs;
    // Update peak RP for register type "i"
    if (liveRegs > peakRegPressures_[i])
      peakRegPressures_[i] = liveRegs;

    // (Chris): Compute sum of live range lengths at this point
    if (GetSpillCostFunc() == SCF_SLIL) {
      sumOfLiveIntervalLengths_[i] += liveRegs_[i].GetOneCnt();
      for (int j = 0; j < liveRegs_[i].GetSize(); ++j) {
        if (liveRegs_[i].GetBit(j)) {
          const Register *reg = regFiles_[i].GetReg(j);
          if (!reg->IsInInterval(inst) && !reg->IsInPossibleInterval(inst)) {
            ++dynamicSlilLowerBound_;
          }
        }
      }
    }

    // FIXME: Can this be taken out of this loop?
    if (GetSpillCostFunc() == SCF_SLIL) {
      //slilSpillCost_ = std::accumulate(sumOfLiveIntervalLengths_,
      //                                 &sumOfLiveIntervalLengths_[regTypeCnt_], 
      //	  		         0);
      accumulator = 0;
      for (int x = 0; x < regTypeCnt_; x++)
        accumulator += sumOfLiveIntervalLengths_[x];
      slilSpillCost_ = accumulator;
    }
  }

  if (GetSpillCostFunc() == SCF_TARGET) {
    //Cannot simply call due to polymorphism, replacing with GenericTarget
    //method for now since I am testing for CPU
    //TODO: Add ability to calculate GCNTarget getCost when compiling for
    //AMD GPU
    //newSpillCost = OST->getCost(regPressures_);
    accumulator = 0;
    for (int x = 0; x < regTypeCnt_; x++)
      accumulator += regPressures_[x];
    newSpillCost = accumulator;

  } else if (GetSpillCostFunc() == SCF_SLIL) {
    //slilSpillCost_ = std::accumulate(sumOfLiveIntervalLengths_,
    //                                 &sumOfLiveIntervalLengths_[regTypeCnt_], 
    //				       0);
    accumulator = 0;
    for (int x = 0; x < regTypeCnt_; x++)
      accumulator += sumOfLiveIntervalLengths_[x];
    slilSpillCost_ = accumulator;

  } else if (GetSpillCostFunc() == SCF_PRP) {
    //newSpillCost =
    //    std::accumulate(regPressures_, &regPressures_[regTypeCnt_], 0);
    accumulator = 0;
    for (int x = 0; x < regTypeCnt_; x++)
      accumulator += regPressures_[x];
    newSpillCost = accumulator;

  } else if (GetSpillCostFunc() == SCF_PEAK_PER_TYPE) {
    for (int i = 0; i < regTypeCnt_; i++)
      if (0 < peakRegPressures_[i] - machMdl_->GetPhysRegCnt(i))
        newSpillCost += peakRegPressures_[i] - machMdl_->GetPhysRegCnt(i);

  } else {
    // Default is PERP (Some SCF like SUM rely on PERP being the default here)
    //int i = 0;
    //std::for_each(
    //    regPressures_, &regPressures_[regTypeCnt_], [&](InstCount RP) {
    //      newSpillCost += std::max(0, RP - machMdl_->GetPhysRegCnt(i++));
    //    });
    for (int i = 0; i < regTypeCnt_; i++) {
      if (0 < regPressures_[i] - machMdl_->GetPhysRegCnt(i))
        newSpillCost += regPressures_[i] - machMdl_->GetPhysRegCnt(i);
    }
  }

#ifdef IS_DEBUG_SLIL_CORRECT
  if (OPTSCHED_gPrintSpills) {
#ifdef __CUDA_ARCH__
    printf("Printing live range lengths for instruction AFTER calculation.\n");
#else
    Logger::Info(
        "Printing live range lengths for instruction AFTER calculation.");
#endif
    for (int j = 0; j < sumOfLiveIntervalLengths_size_; j++) {
#ifdef __CUDA_ARCH__
      printf("SLIL for regType %d is currently %d\n", j,
	     sumOfLiveIntervalLengths_[j]);
#else
      Logger::Info("SLIL for regType %d is currently %d", j,
                   sumOfLiveIntervalLengths_[j]);
#endif
    }
  }
#endif

  crntStepNum_++;
  spillCosts_[crntStepNum_] = newSpillCost;

#ifdef IS_DEBUG_REG_PRESSURE
#ifdef __CUDA_ARCH__
  printf("Spill cost at step  %d = %d\n", crntStepNum_, newSpillCost);
#else
  Logger::Info("Spill cost at step  %d = %d", crntStepNum_, newSpillCost);
#endif
#endif

  totSpillCost_ += newSpillCost;

  //peakSpillCost_ = std::max(peakSpillCost_, newSpillCost);
  if (peakSpillCost_ < newSpillCost)
    peakSpillCost_ = newSpillCost;

  CmputCrntSpillCost_();

  schduldInstCnt_++;
  if (inst->MustBeInBBEntry())
    schduldEntryInstCnt_++;
  if (inst->MustBeInBBExit())
    schduldExitInstCnt_++;
}
/*****************************************************************************/

void BBWithSpill::UpdateSpillInfoForUnSchdul_(SchedInstruction *inst) {
  int16_t regType;
  int i, defCnt, useCnt, regNum, physRegNum;
  Register **defs, **uses;
  Register *def, *use;
  bool isLive;

#ifdef IS_DEBUG_REG_PRESSURE
  Logger::Info("Updating reg pressure after unscheduling Inst %d",
               inst->GetNum());
#endif

  defCnt = inst->GetDefs(defs);
  useCnt = inst->GetUses(uses);

  // (Chris): Update the SLIL for all live regs at this point.
  if (GetSpillCostFunc() == SCF_SLIL) {
    for (int i = 0; i < regTypeCnt_; ++i) {
      for (int j = 0; j < liveRegs_[i].GetSize(); ++j) {
        if (liveRegs_[i].GetBit(j)) {
          const Register *reg = regFiles_[i].GetReg(j);
          sumOfLiveIntervalLengths_[i]--;
          if (!reg->IsInInterval(inst) && !reg->IsInPossibleInterval(inst)) {
            --dynamicSlilLowerBound_;
          }
        }
      }
      assert(sumOfLiveIntervalLengths_[i] >= 0 &&
             "UpdateSpillInfoForUnSchdul_: SLIL negative!");
    }
  }

  // Update Live regs
  for (i = 0; i < defCnt; i++) {
    def = defs[i];
    regType = def->GetType();
    regNum = def->GetNum();
    physRegNum = def->GetPhysicalNumber();

#ifdef IS_DEBUG_REG_PRESSURE
    Logger::Info("Inst %d defines reg %d of type %d and %d uses",
                 inst->GetNum(), regNum, regType, def->GetUseCnt());
#endif

    // if (def->GetUseCnt() > 0) {
    assert(liveRegs_[regType].GetBit(regNum));
    liveRegs_[regType].SetBit(regNum, false, def->GetWght());

#ifdef IS_DEBUG_REG_PRESSURE
    Logger::Info("Reg type %d now has %d live regs", regType,
                 liveRegs_[regType].GetOneCnt());
#endif

    if (regFiles_[regType].GetPhysRegCnt() > 0 && physRegNum >= 0)
      livePhysRegs_[regType].SetBit(physRegNum, false, def->GetWght());
    def->ResetCrntUseCnt();
    //}
  }

  for (i = 0; i < useCnt; i++) {
    use = uses[i];
    regType = use->GetType();
    regNum = use->GetNum();
    physRegNum = use->GetPhysicalNumber();

#ifdef IS_DEBUG_REG_PRESSURE
    Logger::Info("Inst %d uses reg %d of type %d and %d uses", inst->GetNum(),
                 regNum, regType, use->GetUseCnt());
#endif

    isLive = use->IsLive();
    use->DelCrntUse();
    assert(use->IsLive());

    if (isLive == false) {
      // (Chris): Since this was the last use, the above SLIL calculation didn't
      // take this instruction into account.
      if (GetSpillCostFunc() == SCF_SLIL) {
        sumOfLiveIntervalLengths_[regType]--;
        if (!use->IsInInterval(inst) && !use->IsInPossibleInterval(inst)) {
          --dynamicSlilLowerBound_;
        }
        assert(sumOfLiveIntervalLengths_[regType] >= 0 &&
               "UpdateSpillInfoForUnSchdul_: SLIL negative!");
      }
      liveRegs_[regType].SetBit(regNum, true, use->GetWght());

#ifdef IS_DEBUG_REG_PRESSURE
      Logger::Info("Reg type %d now has %d live regs", regType,
                   liveRegs_[regType].GetOneCnt());
#endif

      if (regFiles_[regType].GetPhysRegCnt() > 0 && physRegNum >= 0)
        livePhysRegs_[regType].SetBit(physRegNum, true, use->GetWght());
    }
  }

  schduldInstCnt_--;
  if (inst->MustBeInBBEntry())
    schduldEntryInstCnt_--;
  if (inst->MustBeInBBExit())
    schduldExitInstCnt_--;

  totSpillCost_ -= spillCosts_[crntStepNum_];
  crntStepNum_--;

#ifdef IS_DEBUG_REG_PRESSURE
// Logger::Info("Spill cost at step  %d = %d", crntStepNum_, newSpillCost);
#endif
}
/*****************************************************************************/

void BBWithSpill::SchdulInst(SchedInstruction *inst, InstCount cycleNum,
                             InstCount slotNum, bool trackCnflcts) {
  crntCycleNum_ = cycleNum;
  crntSlotNum_ = slotNum;
  if (inst == NULL)
    return;
  assert(inst != NULL);
  UpdateSpillInfoForSchdul_(inst, trackCnflcts);
}

void BBWithSpill::Dev_SchdulInst(SchedInstruction *inst, InstCount cycleNum,
                             InstCount slotNum, bool trackCnflcts) {
  crntCycleNum_ = cycleNum;
  crntSlotNum_ = slotNum;
  if (inst == NULL)
    return;
  assert(inst != NULL);
  UpdateSpillInfoForSchdul_(inst, trackCnflcts);
}
/*****************************************************************************/

void BBWithSpill::UnschdulInst(SchedInstruction *inst, InstCount cycleNum,
                               InstCount slotNum, EnumTreeNode *trgtNode) {
  if (slotNum == 0) {
    crntCycleNum_ = cycleNum - 1;
    crntSlotNum_ = machMdl_->GetIssueRate() - 1;
  } else {
    crntCycleNum_ = cycleNum;
    crntSlotNum_ = slotNum - 1;
  }

  if (inst == NULL) {
    return;
  }

  UpdateSpillInfoForUnSchdul_(inst);
  peakSpillCost_ = trgtNode->GetPeakSpillCost();
  CmputCrntSpillCost_();
}
/*****************************************************************************/

void BBWithSpill::FinishHurstc_() {

#ifdef IS_DEBUG_BBSPILL_COST
  stats::traceCostLowerBound.Record(costLwrBound_);
  stats::traceHeuristicCost.Record(hurstcCost_);
  stats::traceHeuristicScheduleLength.Record(hurstcSchedLngth_);
#endif
}
/*****************************************************************************/

void BBWithSpill::FinishOptml_() {
#ifdef IS_DEBUG_BBSPILL_COST
  stats::traceOptimalCost.Record(bestCost_);
  stats::traceOptimalScheduleLength.Record(bestSchedLngth_);
#endif
}
/*****************************************************************************/

Enumerator *BBWithSpill::AllocEnumrtr_(Milliseconds timeout) {
  bool enblStallEnum = enblStallEnum_;
  /*  if (!dataDepGraph_->IncludesUnpipelined()) {
      enblStallEnum = false;
    }*/

  enumrtr_ = new LengthCostEnumerator(
      dataDepGraph_, machMdl_, schedUprBound_, GetSigHashSize(),
      GetEnumPriorities(), GetPruningStrategy(), SchedForRPOnly_, enblStallEnum,
      timeout, GetSpillCostFunc(), 0, NULL);

  return enumrtr_;
}
/*****************************************************************************/

FUNC_RESULT BBWithSpill::Enumerate_(Milliseconds startTime,
                                    Milliseconds rgnTimeout,
                                    Milliseconds lngthTimeout) {
  InstCount trgtLngth;
  FUNC_RESULT rslt = RES_SUCCESS;
  int iterCnt = 0;
  int costLwrBound = 0;
  bool timeout = false;

  Milliseconds rgnDeadline, lngthDeadline;
  rgnDeadline =
      (rgnTimeout == INVALID_VALUE) ? INVALID_VALUE : startTime + rgnTimeout;
  lngthDeadline =
      (rgnTimeout == INVALID_VALUE) ? INVALID_VALUE : startTime + lngthTimeout;
  assert(lngthDeadline <= rgnDeadline);

  for (trgtLngth = schedLwrBound_; trgtLngth <= schedUprBound_; trgtLngth++) {
    InitForSchdulng();
    //#ifdef IS_DEBUG_ENUM_ITERS
    Logger::Info("Enumerating at target length %d", trgtLngth);
    //#endif
    rslt = enumrtr_->FindFeasibleSchedule(enumCrntSched_, trgtLngth, this,
                                          costLwrBound, lngthDeadline);
    if (rslt == RES_TIMEOUT)
      timeout = true;
    HandlEnumrtrRslt_(rslt, trgtLngth);

    if (GetBestCost() == 0 || rslt == RES_ERROR ||
        (lngthDeadline == rgnDeadline && rslt == RES_TIMEOUT) ||
        (rslt == RES_SUCCESS && IsSecondPass())) {

      // If doing two pass optsched and on the second pass then terminate if a
      // schedule is found with the same min-RP found in first pass.
      if (rslt == RES_SUCCESS && IsSecondPass()) {
        Logger::Info("Schedule found in second pass, terminating BB loop.");

        if (trgtLngth < schedUprBound_)
          Logger::Info("Schedule found with length %d is shorter than current "
                       "schedule with length %d.",
                       trgtLngth, schedUprBound_);
      }

      break;
    }

    enumrtr_->Reset();
    enumCrntSched_->Reset();

    if (!IsSecondPass())
      CmputSchedUprBound_();

    iterCnt++;
    costLwrBound += 1;
    lngthDeadline = Utilities::GetProcessorTime() + lngthTimeout;
    if (lngthDeadline > rgnDeadline)
      lngthDeadline = rgnDeadline;
  }

#ifdef IS_DEBUG_ITERS
  stats::iterations.Record(iterCnt);
  stats::enumerations.Record(enumrtr_->GetSearchCnt());
  stats::lengths.Record(iterCnt);
#endif

  // Failure to find a feasible sched. in the last iteration is still
  // considered an overall success
  if (rslt == RES_SUCCESS || rslt == RES_FAIL) {
    rslt = RES_SUCCESS;
  }
  if (timeout)
    rslt = RES_TIMEOUT;

  return rslt;
}
/*****************************************************************************/

InstCount BBWithSpill::UpdtOptmlSched(InstSchedule *crntSched,
                                      LengthCostEnumerator *) {
  InstCount crntCost;
  InstCount crntExecCost;

  //  crntCost = CmputNormCost_(crntSched, CCM_DYNMC, crntExecCost, false);
  crntCost = CmputNormCost_(crntSched, CCM_STTC, crntExecCost, false);

  //#ifdef IS_DEBUG_SOLN_DETAILS_2
  Logger::Info(
      "Found a feasible sched. of length %d, spill cost %d and tot cost %d",
      crntSched->GetCrntLngth(), crntSched->GetSpillCost(), crntCost);
  //  crntSched->Print(Logger::GetLogStream(), "New Feasible Schedule");
  //#endif

  if (crntCost < GetBestCost()) {

    if (crntSched->GetCrntLngth() > schedLwrBound_)
      Logger::Info("$$$ GOOD_HIT: Better spill cost for a longer schedule");

    SetBestCost(crntCost);
    optmlSpillCost_ = crntSpillCost_;
    SetBestSchedLength(crntSched->GetCrntLngth());
    enumBestSched_->Copy(crntSched);
    bestSched_ = enumBestSched_;
  }

  return GetBestCost();
}
/*****************************************************************************/

void BBWithSpill::SetupForSchdulng_() {
  for (int i = 0; i < regTypeCnt_; i++) {
    liveRegs_[i].Construct(regFiles_[i].GetRegCnt());
  }

  SetupPhysRegs_();

  entryInstCnt_ = dataDepGraph_->GetEntryInstCnt();
  exitInstCnt_ = dataDepGraph_->GetExitInstCnt();
  schduldEntryInstCnt_ = 0;
  schduldExitInstCnt_ = 0;

  /*
  if (chkCnflcts_)
    for (int i = 0; i < regTypeCnt_; i++) {
      regFiles_[i].SetupConflicts();
    }
 */
}
/*****************************************************************************/

bool BBWithSpill::ChkCostFsblty(InstCount trgtLngth, EnumTreeNode *node) {
  bool fsbl = true;
  InstCount crntCost, dynmcCostLwrBound;
  if (GetSpillCostFunc() == SCF_SLIL) {
    crntCost = dynamicSlilLowerBound_ * SCW_ + trgtLngth * schedCostFactor_;
  } else {
    crntCost = crntSpillCost_ * SCW_ + trgtLngth * schedCostFactor_;
  }
  crntCost -= GetCostLwrBound();
  dynmcCostLwrBound = crntCost;

  // assert(cost >= 0);
  assert(dynmcCostLwrBound >= 0);

  fsbl = dynmcCostLwrBound < GetBestCost();

  // FIXME: RP tracking should be limited to the current SCF. We need RP
  // tracking interface.
  if (fsbl) {
    node->SetCost(crntCost);
    node->SetCostLwrBound(dynmcCostLwrBound);
    node->SetPeakSpillCost(peakSpillCost_);
    node->SetSpillCostSum(totSpillCost_);
  }
  return fsbl;
}
/*****************************************************************************/

void BBWithSpill::SetSttcLwrBounds(EnumTreeNode *) {
  // Nothing.
}

/*****************************************************************************/

__host__ __device__
bool BBWithSpill::ChkInstLglty(SchedInstruction *inst) {
  return true;
  /*
  int16_t regType;
  int defCnt, physRegNum;
  Register **defs;
  Register *def, *liveDef;

#ifdef IS_DEBUG_CHECK
  Logger::Info("Checking inst %d %s", inst->GetNum(), inst->GetOpCode());
#endif

  if (fixLivein_) {
    if (inst->MustBeInBBEntry() == false &&
        schduldEntryInstCnt_ < entryInstCnt_)
      return false;
  }

  if (fixLiveout_) {
    if (inst->MustBeInBBExit() == true &&
        schduldInstCnt_ < (dataDepGraph_->GetInstCnt() - exitInstCnt_))
      return false;
  }

  defCnt = inst->GetDefs(defs);

  // Update Live regs
  for (int i = 0; i < defCnt; i++) {
    def = defs[i];
    regType = def->GetType();
    physRegNum = def->GetPhysicalNumber();

    // If this is a physical register definition and another
    // definition of the same physical register is live, then
    // scheduling this instruction is illegal unless this
    // instruction is the last use of that physical reg definition.
    if (regFiles_[regType].GetPhysRegCnt() > 0 && physRegNum >= 0 &&
        livePhysRegs_[regType].GetBit(physRegNum) == true) {

      liveDef = regFiles_[regType].FindLiveReg(physRegNum);
      assert(liveDef != NULL);

      // If this instruction is the last use of the current live def
      if (liveDef->GetCrntUseCnt() + 1 == liveDef->GetUseCnt() &&
          inst->FindUse(liveDef) == true)
        return true;
      else
        return false;
    } // end if
  }   // end for
  return true;
  */
}

bool BBWithSpill::ChkSchedule_(InstSchedule *bestSched,
                               InstSchedule *lstSched) {
  return true;
  /*
  if (bestSched == NULL || bestSched == lstSched)
    return true;
  if (chkSpillCostSum_) {

    InstCount i, heurLarger = 0, bestLarger = 0;
    for (i = 0; i < dataDepGraph_->GetInstCnt(); i++) {
      if (lstSched->GetSpillCost(i) > bestSched->GetSpillCost(i))
        heurLarger++;
      if (bestSched->GetSpillCost(i) > lstSched->GetSpillCost(i))
        bestLarger++;
    }
    Logger::Info("Heuristic spill cost is larger at %d points, while best "
                 "spill cost is larger at %d points",
                 heurLarger, bestLarger);
    if (bestSched->GetTotSpillCost() > lstSched->GetTotSpillCost()) {
      // Enumerator's best schedule has a greater spill cost sum than the
      // heuristic
      // This can happen if we are using a cost function other than the spill
      // cost sum function
      Logger::Info("??? Heuristic sched has a smaller spill cost sum than best "
                   "sched, heur : %d, best : %d. ",
                   lstSched->GetTotSpillCost(), bestSched->GetTotSpillCost());
      if (lstSched->GetCrntLngth() <= bestSched->GetCrntLngth()) {
        Logger::Info("Taking heuristic schedule");
        bestSched->Copy(lstSched);
        return false;
      }
    }
  }
  if (chkCnflcts_) {
    CmputCnflcts_(lstSched);
    CmputCnflcts_(bestSched);

#ifdef IS_DEBUG_CONFLICTS
    Logger::Info("Heuristic conflicts : %d, best conflicts : %d. ",
                 lstSched->GetConflictCount(), bestSched->GetConflictCount());
#endif

    if (bestSched->GetConflictCount() > lstSched->GetConflictCount()) {
      // Enumerator's best schedule causes more conflicst than the heuristic
      // schedule.
      Logger::Info("??? Heuristic sched causes fewer conflicts than best "
                   "sched, heur : %d, best : %d. ",
                   lstSched->GetConflictCount(), bestSched->GetConflictCount());
      if (lstSched->GetCrntLngth() <= bestSched->GetCrntLngth()) {
        Logger::Info("Taking heuristic schedule");
        bestSched->Copy(lstSched);
        return false;
      }
    }
  }
  return true;
  */
}

void BBWithSpill::CmputCnflcts_(InstSchedule *sched) {
  int cnflctCnt = 0;
  InstCount execCost;

  CmputNormCost_(sched, CCM_STTC, execCost, true);
  for (int i = 0; i < regTypeCnt_; i++) {
    cnflctCnt += regFiles_[i].GetConflictCnt();
  }
  sched->SetConflictCount(cnflctCnt);
}

void BBWithSpill::CopyPointersToDevice(SchedRegion* dev_rgn) {
  //copy peakRegPressures_ to device
  InstCount *dev_peakRegPressures = NULL;

  //allocate device mem
  if (hipSuccess != hipMalloc((void**)&dev_peakRegPressures, regTypeCnt_ * sizeof(InstCount)))
    printf("Error allocating dev mem for dev_peakRegPressures: %s\n", hipGetErrorString(hipGetLastError()));

  //copy array to device
  if (hipSuccess != hipMemcpy(dev_peakRegPressures, peakRegPressures_, regTypeCnt_ * sizeof(InstCount), hipMemcpyHostToDevice))
    printf("Error copying peakRegPressures to device: %s\n", hipGetErrorString(hipGetLastError()));

  //update device pointer
  if (hipSuccess != hipMemcpy(&(((BBWithSpill *)dev_rgn)->peakRegPressures_), &dev_peakRegPressures, sizeof(InstCount *), hipMemcpyHostToDevice))
    printf("Error updating dev_rgn->peakRegPressures_: %s\n", hipGetErrorString(hipGetLastError()));

  //copy spillCosts_ to device
  InstCount *dev_spillCosts = NULL;

  //allocate dev mem
  if (hipSuccess != hipMalloc((void**)&dev_spillCosts, dataDepGraph_->GetInstCnt() * sizeof(InstCount)))
    printf("Error allocating dev mem for dev_spillCosts: %s\n", hipGetErrorString(hipGetLastError()));

  //copy array to device
  if (hipSuccess != hipMemcpy(dev_spillCosts, spillCosts_, dataDepGraph_->GetInstCnt() * sizeof(InstCount), hipMemcpyHostToDevice))
    printf("Error copying spillCosts_ to device: %s\n", hipGetErrorString(hipGetLastError()));

  //update device pointer
  if (hipSuccess != hipMemcpy(&(((BBWithSpill *)dev_rgn)->spillCosts_), &dev_spillCosts, sizeof(InstCount *), hipMemcpyHostToDevice))
    printf("Error updating dev_rgn->spillCosts_: %s\n", hipGetErrorString(hipGetLastError()));

  //copy liveRegs to device
  WeightedBitVector *dev_liveRegs = NULL;

  //allocate dev mem
  if (hipSuccess != hipMalloc((void**)&dev_liveRegs, regTypeCnt_ * sizeof(WeightedBitVector)))
    printf("Error allocating dev mem for dev_liveRegs: %s\n", hipGetErrorString(hipGetLastError()));

  //copy array
  if (hipSuccess != hipMemcpy(dev_liveRegs, liveRegs_, regTypeCnt_ * sizeof(WeightedBitVector), hipMemcpyHostToDevice))
    printf("Error copying liveRegs_ to device: %s\n", hipGetErrorString(hipGetLastError()));

  //update device pointer
  if (hipSuccess != hipMemcpy(&(((BBWithSpill *)dev_rgn)->liveRegs_), &dev_liveRegs, sizeof(WeightedBitVector *), hipMemcpyHostToDevice))
    printf("Error updating dev_rng->liveRegs_ on device: %s\n", hipGetErrorString(hipGetLastError()));

  //Copy pointers in each liveRegs_[i] to device
  unsigned int *vctr = NULL;
  unsigned int *dev_vctr = NULL;
  int unitCnt;
  for (int i = 0; i < regTypeCnt_; i++) {
    vctr = liveRegs_[i].GetVctrCpy();
    unitCnt = liveRegs_[i].GetUnitCnt();

    //allocate device mem
    if (hipSuccess != hipMalloc((void**)&dev_vctr, unitCnt * sizeof(unsigned int)))
      printf("Error allocating dev mem for dev_vctr: %s\n", hipGetErrorString(hipGetLastError()));

    //copy vctr to device
    if (hipSuccess != hipMemcpy(dev_vctr, vctr, unitCnt * sizeof(unsigned int), hipMemcpyHostToDevice))
      printf("Error copying vctr to device: %s\n", hipGetErrorString(hipGetLastError()));

    //update device pointer
    if (hipSuccess != hipMemcpy(&(dev_liveRegs[i].vctr_), &dev_vctr, sizeof(unsigned int *), hipMemcpyHostToDevice))
      printf("Error updating dev_liveRegs[%d].vctr_: %s\n", i, hipGetErrorString(hipGetLastError()));

    //delete vctr copy
    delete[] vctr;
  }

  //copy liveRegs to device
  WeightedBitVector *dev_livePhysRegs = NULL;

  //allocate dev mem
  if (hipSuccess != hipMalloc((void**)&dev_livePhysRegs, regTypeCnt_ * sizeof(WeightedBitVector)))
    printf("Error allocating dev mem for dev_livePhysRegs: %s\n", hipGetErrorString(hipGetLastError()));

  //copy array
  if (hipSuccess != hipMemcpy(dev_livePhysRegs, livePhysRegs_, regTypeCnt_ * sizeof(WeightedBitVector), hipMemcpyHostToDevice))
    printf("Error copying livePhysRegs_ to device: %s\n", hipGetErrorString(hipGetLastError()));

  //update device pointer
  if (hipSuccess != hipMemcpy(&(((BBWithSpill *)dev_rgn)->livePhysRegs_), &dev_livePhysRegs, sizeof(WeightedBitVector *), hipMemcpyHostToDevice))
    printf("Error updating dev_rng->livePhysRegs_ on device: %s\n", hipGetErrorString(hipGetLastError()));

  //Copy pointers in each livePhysRegs_[i] to device
  for (int i = 0; i < regTypeCnt_; i++) {
    vctr = livePhysRegs_[i].GetVctrCpy();
    unitCnt = livePhysRegs_[i].GetUnitCnt();

    //allocate device mem
    if (hipSuccess != hipMalloc((void**)&dev_vctr, unitCnt * sizeof(unsigned int)))
      printf("Error allocating dev mem for dev_vctr: %s\n", hipGetErrorString(hipGetLastError()));

    //copy vctr to device
    if (hipSuccess != hipMemcpy(dev_vctr, vctr, unitCnt * sizeof(unsigned int), hipMemcpyHostToDevice))
      printf("Error copying vctr to device: %s\n", hipGetErrorString(hipGetLastError()));

    //update device pointer
    if (hipSuccess != hipMemcpy(&(dev_livePhysRegs[i].vctr_), &dev_vctr, sizeof(unsigned int *), hipMemcpyHostToDevice))
      printf("Error updating dev_livePhysRegs[%d].vctr_: %s\n", i, hipGetErrorString(hipGetLastError()));

    //Delete vctr copy
    delete[] vctr;
  }

  //copy sumOfLiveIntervalLengths to device
  int *dev_SLIL = NULL;

  //allocate device memory
  if (hipSuccess != hipMalloc((void**)&dev_SLIL, regTypeCnt_ * sizeof(int)))
    printf("Error allocating dev mem for dev_SLIL: %s\n", hipGetErrorString(hipGetLastError()));

  //copy sumOfLiveIntervalLengths to device
  if (hipSuccess != hipMemcpy(dev_SLIL, sumOfLiveIntervalLengths_, regTypeCnt_ * sizeof(int), hipMemcpyHostToDevice))
    printf("Error copying SLIL to device: %s\n", hipGetErrorString(hipGetLastError()));

  //update device pointer dev_rgn->sumOfLiveIntervalLengths
  if (hipSuccess != hipMemcpy(&(((BBWithSpill *)dev_rgn)->sumOfLiveIntervalLengths_), &dev_SLIL, sizeof(int *), hipMemcpyHostToDevice))
    printf("Error updating dev_rgn->SLIL: %s\n", hipGetErrorString(hipGetLastError()));

  //copy regPressures_ to device
  unsigned *dev_regPressures = NULL;

  //allocate device memry
  if (hipSuccess != hipMalloc((void**)&dev_regPressures, regTypeCnt_ * sizeof(unsigned)))
    printf("Error allocating dev mem for dev_regPressures: %s\n", hipGetErrorString(hipGetLastError()));

  //copy regPressures_ to device
  if (hipSuccess != hipMemcpy(dev_regPressures, regPressures_, regTypeCnt_ * sizeof(unsigned), hipMemcpyHostToDevice))
    printf("Error copying regPressures_ to device: %s\n", hipGetErrorString(hipGetLastError()));

  //update device pointer dev_rgn->sumOfLiveIntervalLengths
  if (hipSuccess != hipMemcpy(&(((BBWithSpill *)dev_rgn)->regPressures_), &dev_regPressures, sizeof(unsigned *), hipMemcpyHostToDevice))
    printf("Error updating dev_rgn->regPressures_: %s\n", hipGetErrorString(hipGetLastError()));
  
  //copy RegFiles_ to device
  RegisterFile *dev_regFiles = NULL;

  //allocate device memory
  if (hipSuccess != hipMallocManaged((void**)&dev_regFiles, machMdl_->GetRegTypeCnt() * sizeof(RegisterFile)))
    printf("Error allocating dev mem for dev_regFiles: %s\n", hipGetErrorString(hipGetLastError()));

  //copy RegFiles_ to device
  if (hipSuccess != hipMemcpy(dev_regFiles, regFiles_, machMdl_->GetRegTypeCnt() * sizeof(RegisterFile), hipMemcpyHostToDevice))
    printf("Error copying regPressures_ to device: %s\n", hipGetErrorString(hipGetLastError()));

  //copy each regFiles_' pointers to device
  for (int i = 0; i < machMdl_->GetRegTypeCnt(); i++)
    regFiles_[i].CopyPointersToDevice(&dev_regFiles[i]);

  //update device pointer dev_rgn->regFiles_
  if (hipSuccess != hipMemcpy(&(((BBWithSpill *)dev_rgn)->regFiles_), &dev_regFiles, sizeof(RegisterFile *), hipMemcpyHostToDevice))
    printf("Error updating dev_rgn->regFiles_: %s\n", hipGetErrorString(hipGetLastError()));

  printf("Finished copying BBWithSpill!\n");
}
