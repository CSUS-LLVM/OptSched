#include "opt-sched/Scheduler/register.h"
#include "llvm/ADT/STLExtras.h"

using namespace llvm::opt_sched;

__host__ __device__
int16_t Register::GetType() const { return type_; }

__device__ __host__
int Register::GetNum() const { return num_; }

__host__ __device__
int Register::GetWght() const { return wght_; }

void Register::SetType(int16_t type) { type_ = type; }

void Register::SetNum(int num) { num_ = num; }

void Register::SetWght(int wght) { wght_ = wght; }

bool Register::IsPhysical() const { return physicalNumber_ != INVALID_VALUE; }

__host__ __device__
int Register::GetPhysicalNumber() const { return physicalNumber_; }

void Register::SetPhysicalNumber(int physicalNumber) {
  physicalNumber_ = physicalNumber;
}

__host__ __device__
bool Register::IsLive() const {
  assert(crntUseCnt_ <= useCnt_);
  return crntUseCnt_ < useCnt_;
}

bool Register::IsLiveIn() const { return liveIn_; }

__host__ __device__
bool Register::IsLiveOut() const { return liveOut_; }

void Register::SetIsLiveIn(bool liveIn) { liveIn_ = liveIn; }

void Register::SetIsLiveOut(bool liveOut) { liveOut_ = liveOut; }

__host__ __device__
void Register::ResetCrntUseCnt() { crntUseCnt_ = 0; }

void Register::AddUse(const SchedInstruction *inst) {
  uses_.insert(inst->GetNum());
  useCnt_++;
}

void Register::AddDef(const SchedInstruction *inst) {
  defs_.insert(inst->GetNum());
  defCnt_++;
}

__device__ __host__
int Register::GetUseCnt() const { return useCnt_; }

const Register::InstSetType &Register::GetUseList() const { return uses_; }

size_t Register::GetSizeOfUseList() const { return uses_.size(); }

int Register::GetDefCnt() const { return defCnt_; }

const Register::InstSetType &Register::GetDefList() const { return defs_; }

size_t Register::GetSizeOfDefList() const { return defs_.size(); }

__device__
void Register::ResetDefsAndUses() {
  // Only called on device, but will not compile without
  // macro since SmallPtrSet doesnt have a Reset() method
  defs_.Reset();
  defCnt_ = 0;
  uses_.Reset();
  useCnt_ = 0;
}

__device__ __host__
int Register::GetCrntUseCnt() const { return crntUseCnt_; }

__host__ __device__
void Register::AddCrntUse() { crntUseCnt_++; }

void Register::DelCrntUse() { crntUseCnt_--; }

__host__ __device__
void Register::ResetCrntLngth() { crntLngth_ = 0; }

int Register::GetCrntLngth() const { return crntLngth_; }

void Register::IncrmntCrntLngth() { crntLngth_++; }

void Register::DcrmntCrntLngth() { crntLngth_--; }

Register &Register::operator=(const Register &rhs) {
  if (this != &rhs) {
    num_ = rhs.num_;
    type_ = rhs.type_;
  }

  return *this;
}

void Register::SetupConflicts(int regCnt) { conflicts_.Construct(regCnt); }

__host__ __device__
void Register::ResetConflicts() {
  conflicts_.Reset();
  isSpillCnddt_ = false;
}

__host__ __device__
void Register::AddConflict(int regNum, bool isSpillCnddt) {
  assert(regNum != num_);
  assert(regNum >= 0);
  conflicts_.SetBit(regNum, true);
  isSpillCnddt_ = isSpillCnddt_ || isSpillCnddt;
}

int Register::GetConflictCnt() const { return conflicts_.GetOneCnt(); }

bool Register::IsSpillCandidate() const { return isSpillCnddt_; }

bool Register::AddToInterval(const SchedInstruction *inst) {
//#ifdef __CUDA_ARCH__
  return liveIntervalSet_.insert(inst->GetNum());
//#else
  //return liveIntervalSet_.insert(inst->GetNum()).second;
//#endif
}

__host__ __device__
bool Register::IsInInterval(const SchedInstruction *inst) const {
//#ifdef __CUDA_ARCH__
  return liveIntervalSet_.contains(inst->GetNum());
//#else
  //return liveIntervalSet_.count(inst->GetNum()) != 0;
//#endif
}

const Register::InstSetType &Register::GetLiveInterval() const {
  return liveIntervalSet_;
}

bool Register::AddToPossibleInterval(const SchedInstruction *inst) {
//#ifdef __CUDA_ARCH__
  return possibleLiveIntervalSet_.insert(inst->GetNum());
//#else
  //return possibleLiveIntervalSet_.insert(inst->GetNum()).second;
//#endif
}

__host__ __device__
bool Register::IsInPossibleInterval(const SchedInstruction *inst) const {
//#ifdef __CUDA_ARCH__
  return possibleLiveIntervalSet_.contains(inst->GetNum());
//#else
  //return possibleLiveIntervalSet_.count(inst->GetNum()) != 0;
//#endif
}

const Register::InstSetType &Register::GetPossibleLiveInterval() const {
  return possibleLiveIntervalSet_;
}

__device__
void Register::ResetLiveIntervals() {
  // Only called on device, but will not compile without
  // macro since SmallPtrSet doesnt have a Reset() method
//#ifdef __CUDA_ARCH__
  liveIntervalSet_.Reset();
  possibleLiveIntervalSet_.Reset();
//#endif
}

void Register::CopyPointersToDevice(Register *dev_reg) {
  size_t memSize;
  // Copy conflicts->vctr_ to device
  unsigned long *dev_vctr;
  if (conflicts_.GetUnitCnt() > 0) {
    memSize = sizeof(unsigned long) * conflicts_.GetUnitCnt();
    if (hipSuccess != hipMalloc(&dev_vctr, memSize))
      Logger::Fatal("Failed to alloc dev mem for conflicts_->vctr");

    if (hipSuccess != hipMemcpy(dev_vctr, conflicts_.vctr_, memSize,
                                  hipMemcpyHostToDevice))
      Logger::Fatal("Failed to copy conflicts_.vctr to device");

    if (hipSuccess != hipMemcpy(&dev_reg->conflicts_.vctr_, &dev_vctr,
                                  sizeof(unsigned long *),
                                  hipMemcpyHostToDevice))
      Logger::Fatal("Failed to update conflicts_->vctr");
  }

  // Copy uses_.elmnt array
  InstCount *dev_elmnt;
  if (uses_.alloc_ > 0) {
    memSize = sizeof(InstCount) * uses_.alloc_;
    if (hipSuccess != hipMalloc(&dev_elmnt, memSize))
      Logger::Fatal("Failed to alloc dev mem for dev_uses.elmnt");

    if (hipSuccess != hipMemcpy(dev_elmnt, uses_.elmnt, memSize,
			          hipMemcpyHostToDevice))
      Logger::Fatal("Failed to copy uses_.elmnt to dev");

    if (hipSuccess != hipMemcpy(&dev_reg->uses_.elmnt, &dev_elmnt,
			          sizeof(InstCount *), hipMemcpyHostToDevice))
      Logger::Fatal("Failed to update dev pointers for uses.elmnt");
  }

  // Copy defs_.elmnt array
  if (defs_.alloc_ > 0) {
    memSize = sizeof(InstCount) * defs_.alloc_;
    if (hipSuccess != hipMalloc(&dev_elmnt, memSize))
      Logger::Fatal("Failed to alloc dev mem for dev_defs.elmnt");

    if (hipSuccess != hipMemcpy(dev_elmnt, defs_.elmnt, memSize,
                                  hipMemcpyHostToDevice))
      Logger::Fatal("Failed to copy defs_.elmnt to dev");

    if (hipSuccess != hipMemcpy(&dev_reg->defs_.elmnt, &dev_elmnt,
                                  sizeof(InstCount *), hipMemcpyHostToDevice))
      Logger::Fatal("Failed to update dev pointers for defs.elmnt");
  }

  // Copy liveIntervalSet_.elmnt array
  if (liveIntervalSet_.alloc_ > 0) {
    memSize = sizeof(InstCount) * liveIntervalSet_.alloc_;
    if (hipSuccess != hipMalloc(&dev_elmnt, memSize))
      Logger::Fatal("Failed to alloc dev mem for dev_liveIntervalSet.elmnt");

    if (hipSuccess != hipMemcpy(dev_elmnt, liveIntervalSet_.elmnt, memSize,
                                  hipMemcpyHostToDevice))
      Logger::Fatal("Failed to copy liveIntervalSet_.elmnt to dev");

    if (hipSuccess != hipMemcpy(&dev_reg->liveIntervalSet_.elmnt, &dev_elmnt,
                                  sizeof(InstCount *), hipMemcpyHostToDevice))
      Logger::Fatal("Failed to update dev pointers for liveIntervalSet.elmnt");
  }

  // Copy possibleLiveIntervalSet_.elmnt array
  if (possibleLiveIntervalSet_.alloc_ > 0) {
    memSize = sizeof(InstCount) * possibleLiveIntervalSet_.alloc_;
    if (hipSuccess != hipMalloc(&dev_elmnt, memSize))
      Logger::Fatal("Failed to alloc dev mem for possibleLiveIntervalSet.elmnt");

    if (hipSuccess != hipMemcpy(dev_elmnt, possibleLiveIntervalSet_.elmnt, 
			          memSize, hipMemcpyHostToDevice))
      Logger::Fatal("Failed to copy possibleLiveIntervalSet_.elmnt to dev");

    if (hipSuccess != hipMemcpy(&dev_reg->possibleLiveIntervalSet_.elmnt, 
			          &dev_elmnt, sizeof(InstCount *), 
				  hipMemcpyHostToDevice))
      Logger::Fatal("Failed to updt dev ptr for possibleLiveIntervalSet.elmnt");
  }
}

__host__ __device__
Register::Register(int16_t type, int num, int physicalNumber) {
  type_ = type;
  num_ = num;
  wght_ = 1;
  defCnt_ = 0;
  useCnt_ = 0;
  crntUseCnt_ = 0;
  physicalNumber_ = physicalNumber;
  isSpillCnddt_ = false;
  liveIn_ = false;
  liveOut_ = false;
}

__host__ __device__
RegisterFile::RegisterFile() {
  regType_ = 0;
  Regs = NULL;
  Regs_size_ = Regs_alloc_ = 0;
  physRegCnt_ = 0;
}

__host__ __device__
RegisterFile::~RegisterFile() {
  if (Regs) {
    for (int i = 0; i < Regs_size_; i++)
      delete Regs[i];
    delete[] Regs;
  }
}

__host__ __device__
int RegisterFile::GetRegCnt() const { return getCount(); }

__host__ __device__
int16_t RegisterFile::GetRegType() const { return regType_; }

__host__ __device__
void RegisterFile::SetRegType(int16_t regType) { regType_ = regType; }

__host__ __device__
void RegisterFile::ResetCrntUseCnts() {
  for (int i = 0; i < getCount(); i++) {
    Regs[i]->ResetCrntUseCnt();
  }
}

__host__ __device__
void RegisterFile::ResetCrntLngths() {
  for (int i = 0; i < getCount(); i++) {
    Regs[i]->ResetCrntLngth();
  }
}

Register *RegisterFile::getNext() {
  size_t RegNum = Regs_size_;
  Register *Reg = new Register;
  Reg->SetType(regType_);
  Reg->SetNum(RegNum);
  //Regs.push_back(std::move(Reg));

  if (Regs_alloc_ == Regs_size_) {
    if (Regs_alloc_ > 0)
      Regs_alloc_ *= 2;
    else
      Regs_alloc_ = 2;

    Register **resized = new Register *[Regs_alloc_];
    //copy contents of old array
    for (int i = 0; i < Regs_size_; i++)
      resized[i] = Regs[i];
    
    delete[] Regs;
    Regs = resized;
  }
  Regs[Regs_size_++] = std::move(Reg);

  return Regs[RegNum];
}

__host__ __device__
void RegisterFile::SetRegCnt(int regCnt) {
  if (regCnt == 0)
    return;

  //Regs.resize(regCnt);
  if (Regs_size_ > 0)
    delete[] Regs;
  Regs_size_ = Regs_alloc_ = regCnt;
  Regs = new Register *[Regs_alloc_];

  for (int i = 0; i < getCount(); i++) {
    Register *Reg = new Register;
    Reg->SetType(regType_);
    Reg->SetNum(i);
    Regs[i] = Reg;
  }
}

__host__ __device__
Register *RegisterFile::GetReg(int num) const {
  if (num >= 0 && num < getCount()) {
    return Regs[num];
  } else {
    return NULL;
  }
}

Register *RegisterFile::FindLiveReg(int physNum) const {
  for (int i = 0; i < getCount(); i++) {
    if (Regs[i]->GetPhysicalNumber() == physNum && Regs[i]->IsLive() == true)
      return Regs[i];
  }
  return NULL;
}


int RegisterFile::FindPhysRegCnt() {
  int maxPhysNum = -1;
  for (int i = 0; i < getCount(); i++) {
    if (Regs[i]->GetPhysicalNumber() != INVALID_VALUE &&
        Regs[i]->GetPhysicalNumber() > maxPhysNum)
      maxPhysNum = Regs[i]->GetPhysicalNumber();
  }

  // Assume that physical registers are given sequential numbers
  // starting from 0.
  physRegCnt_ = maxPhysNum + 1;
  return physRegCnt_;
}

__host__ __device__
int RegisterFile::GetPhysRegCnt() const { return physRegCnt_; }

void RegisterFile::SetupConflicts() {
  for (int i = 0; i < getCount(); i++)
    Regs[i]->SetupConflicts(getCount());
}

__host__ __device__
void RegisterFile::ResetConflicts() {
  for (int i = 0; i < getCount(); i++)
    Regs[i]->ResetConflicts();
}

int RegisterFile::GetConflictCnt() {
  int cnflctCnt = 0;
  for (int i = 0; i < getCount(); i++) {
    cnflctCnt += Regs[i]->GetConflictCnt();
  }
  return cnflctCnt;
}

__host__ __device__
void RegisterFile::AddConflictsWithLiveRegs(int regNum, int liveRegCnt) {
  bool isSpillCnddt = (liveRegCnt + 1) > physRegCnt_;
  int conflictCnt = 0;
  for (int i = 0; i < getCount(); i++) {
    if (i != regNum && Regs[i]->IsLive() == true) {
      Regs[i]->AddConflict(regNum, isSpillCnddt);
      Regs[regNum]->AddConflict(i, isSpillCnddt);
      conflictCnt++;
    }
    if (conflictCnt == liveRegCnt)
      break;
  }
}

__device__
void RegisterFile::Reset() {
  ResetConflicts();
  ResetCrntUseCnts();
  ResetCrntLngths();
  
  for (int i = 0; i < getCount(); i++) {
    Regs[i]->ResetDefsAndUses();
    Regs[i]->ResetLiveIntervals();
  }
}

void RegisterFile::CopyPointersToDevice(RegisterFile *dev_regFile) {
  //remove reference to host pointer
  dev_regFile->Regs = NULL;
  //declare and allocate array of pointers
  Register **dev_regs = NULL;
  size_t memSize;
  //allocate device memory
  memSize = getCount() * sizeof(Register *);
  if (hipSuccess != hipMalloc((void**)&dev_regs, memSize))
    printf("Error allocating dev mem for dev_regs: %s\n", 
		    hipGetErrorString(hipGetLastError()));

  //copy array of host pointers to device
  if (hipSuccess != hipMemcpy(dev_regs, Regs, memSize, 
			        hipMemcpyHostToDevice))
    printf("Error copying Regs to device: %s\n", 
		    hipGetErrorString(hipGetLastError()));

  //copy each register to device and update its pointer in dev_regs
  Register *dev_reg = NULL;

  for (int i = 0; i < getCount(); i++) {
    //allocate device memory
    if (hipSuccess != hipMalloc((void**)&dev_reg, sizeof(Register)))
      printf("Error allocating dev mem for dev_reg: %s\n", 
		      hipGetErrorString(hipGetLastError()));

    //copy register to device
    if (hipSuccess != hipMemcpy(dev_reg, Regs[i], sizeof(Register), 
			          hipMemcpyHostToDevice))
      printf("Error copying Regs[%d] to device: %s\n", i, 
		      hipGetErrorString(hipGetLastError()));

    Regs[i]->CopyPointersToDevice(dev_reg);

    //update dev_regs pointer
    if (hipSuccess != hipMemcpy(&dev_regs[i], &dev_reg, sizeof(Register *), 
			          hipMemcpyHostToDevice))
      printf("Error updating dev_regs[%d] on device: %s\n", i, 
		      hipGetErrorString(hipGetLastError()));
  }

  //update dev_regFile->Regs pointer
  dev_regFile->Regs = dev_regs;
}
