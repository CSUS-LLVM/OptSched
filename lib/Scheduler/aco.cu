#include "hip/hip_runtime.h"
#include "opt-sched/Scheduler/aco.h"
#include "opt-sched/Scheduler/config.h"
#include "opt-sched/Scheduler/data_dep.h"
#include "opt-sched/Scheduler/random.h"
#include "opt-sched/Scheduler/ready_list.h"
#include "opt-sched/Scheduler/register.h"
#include "opt-sched/Scheduler/sched_region.h"
#include "opt-sched/Scheduler/bb_spill.h"
#include "opt-sched/Scheduler/dev_defines.h"
#include <thrust/functional.h>
#include <hip/hip_cooperative_groups.h>
#include "llvm/ADT/STLExtras.h"
#include <iomanip>
#include <iostream>
#include <sstream>

using namespace llvm::opt_sched;
namespace cg = cooperative_groups;

#ifndef NDEBUG
static void PrintInstruction(SchedInstruction *inst);
#endif
void PrintSchedule(InstSchedule *schedule);

double RandDouble(double min, double max) {
  double rand = (double)RandomGen::GetRand32() / INT32_MAX;
  return (rand * (max - min)) + min;
}

#define USE_ACS 0
#define TWO_STEP 1
#define MIN_DEPOSITION 1
#define MAX_DEPOSITION 6
#define MAX_DEPOSITION_MINUS_MIN (MAX_DEPOSITION - MIN_DEPOSITION)
#define ACO_SCHED_STALLS 1
//#define BIASED_CHOICES 10000000
//#define LOCAL_DECAY 0.1

//#if USE_ACS
//#define ANTS_PER_ITERATION 10
//#define DECAY_FACTOR 0.1
//#else
//#define ANTS_PER_ITERATION count_
//#define DECAY_FACTOR 0.5
//#endif

ACOScheduler::ACOScheduler(DataDepGraph *dataDepGraph,
                           MachineModel *machineModel, InstCount upperBound,
                           SchedPriorities priorities, bool vrfySched, 
                           bool IsPostBB, SchedRegion *dev_rgn,
                           DataDepGraph *dev_DDG, 
                           DeviceVector<Choice> *dev_ready,
			   MachineModel *dev_MM, hiprandState_t *dev_states)
    : ConstrainedScheduler(dataDepGraph, machineModel, upperBound) {
  VrfySched_ = vrfySched;
  this->IsPostBB = IsPostBB;
  prirts_ = priorities;
  rdyLst_ = new ReadyList(dataDepGraph_, priorities);
  count_ = dataDepGraph->GetInstCnt();
  Config &schedIni = SchedulerOptions::getInstance();
  dev_rgn_ = dev_rgn;
  dev_DDG_ = dev_DDG;
  dev_ready_ = dev_ready;
  dev_MM_ = dev_MM;
  dev_states_ = dev_states;
  dev_pheromone_elmnts_alloced_ = false;
  numAntsTerminated_ = 0;

  use_fixed_bias = schedIni.GetBool("ACO_USE_FIXED_BIAS");
  use_tournament = schedIni.GetBool("ACO_TOURNAMENT");
  bias_ratio = schedIni.GetFloat("ACO_BIAS_RATIO");
  local_decay = schedIni.GetFloat("ACO_LOCAL_DECAY");
  decay_factor = schedIni.GetFloat("ACO_DECAY_FACTOR");
  ants_per_iteration = schedIni.GetInt("ACO_ANT_PER_ITERATION");
  print_aco_trace = schedIni.GetBool("ACO_TRACE");
  IsTwoPassEn = schedIni.GetBool("USE_TWO_PASS");

  /*
  std::cerr << "useOldAlg===="<<useOldAlg<<"\n\n";
  std::cerr << "heuristicImportance_===="<<heuristicImportance_<<"\n\n";
  std::cerr << "tournament===="<<tournament<<"\n\n";
  std::cerr << "bias_ratio===="<<bias_ratio<<"\n\n";
  std::cerr << "local_decay===="<<local_decay<<"\n\n";
  std::cerr << "decay_factor===="<<decay_factor<<"\n\n";
  std::cerr << "ants_per_iteration===="<<ants_per_iteration<<"\n\n";
  */

  int pheromone_size = (count_ + 1) * count_;
  pheromone_.resize(pheromone_size);
  InitialSchedule = nullptr;
}

ACOScheduler::~ACOScheduler() { delete rdyLst_; }

// Pheromone table lookup
// -1 means no instruction, so e.g. pheromone(-1, 10) gives pheromone on path
// from empty schedule to schedule only containing instruction 10
__host__ __device__
pheromone_t &ACOScheduler::Pheromone(SchedInstruction *from,
                                     SchedInstruction *to) {
  assert(to != NULL);
  int fromNum = -1;
  if (from != NULL)
    fromNum = from->GetNum();
  return Pheromone(fromNum, to->GetNum());
}

__host__ __device__
pheromone_t &ACOScheduler::Pheromone(InstCount from, InstCount to) {
  int row = 0;
  if (from != -1)
    row = from + 1;
  return pheromone_[(row * count_) + to];
}

__host__ __device__
double ACOScheduler::Score(SchedInstruction *from, Choice choice) {
  return Pheromone(from, choice.inst) *
         pow(choice.heuristic, heuristicImportance_);
}

__host__ __device__
bool ACOScheduler::shouldReplaceSchedule(InstSchedule *OldSched,
                                         InstSchedule *NewSched,
                                         bool IsGlobal) {
  // return true if the old schedule is null (eg:there is no old schedule)
  // return false if the new schedule is is NULL
  if (!NewSched) {
    return false;
  } else if (!OldSched) {
    return true;
  }

  // return false if new schedule is invalid
  // return true if old schedule is invalid
  if (NewSched->GetCost() == INVALID_VALUE)
    return false;
  if (OldSched->GetCost() == INVALID_VALUE)
    return true;

  // if it is the 1st pass return the cost comparison
  // if it is the 2nd pass return true if the RP cost and ILP cost is less
#ifdef __CUDA_ARCH__
  bool isSecondPass = dev_rgn_->IsSecondPass();
#else
  bool isSecondPass = rgn_->IsSecondPass(); 
#endif
  if (!IsTwoPassEn || !isSecondPass) {
    InstCount NewCost = (!IsTwoPassEn) ? NewSched->GetCost() : NewSched->GetNormSpillCost();
    InstCount OldCost = (!IsTwoPassEn) ? OldSched->GetCost() : OldSched->GetNormSpillCost();

    if (NewCost < OldCost)
      return true;
    else
      return false;
  }
  else {
    InstCount NewCost = NewSched->GetExecCost();
    InstCount OldCost = OldSched->GetExecCost();
    InstCount NewSpillCost = NewSched->GetNormSpillCost();
    InstCount OldSpillCost = OldSched->GetNormSpillCost();
    // Lower Spill Cost always wins
    if (NewSpillCost < OldSpillCost)
      return true;
    else if (NewSpillCost == OldSpillCost && NewCost < OldCost)
      return true;
    else
      return false;
  }
}

__host__ __device__
Choice ACOScheduler::SelectInstruction(DeviceVector<Choice> &ready,
                                       SchedInstruction *lastInst,
                                       double ScoreSum) {


  //genereate the random numbers that we will need for deciding if
  //we are going to use the fixed bias or if we are going to use
  //fitness porportional selection.  Generate the number used for
  //the fitness porportional selection point
  double rand;
  pheromone_t point;
#ifdef __CUDA_ARCH__
  rand = hiprand_uniform(&dev_states_[GLOBALTID]);
  point = ScoreSum * hiprand_uniform(&dev_states_[GLOBALTID]);
#else
  rand = RandDouble(0, 1);
  point = RandDouble(0, ScoreSum);
#endif

  //here we compute the chance that we will use fp selection or auto pick the best
  double choose_best_chance;
  if (use_fixed_bias) { //this is a non-diverging if stmt
    choose_best_chance = (1 - (double)fixed_bias / count_) * (0 < 1 - (double)fixed_bias / count_);
  } else
    choose_best_chance = bias_ratio;


  //here we determine the max scoring instruction and the fp choice
  //this code is a bit dense, but what we are doing is picking the
  //indices of the max and fp choice instructions
  //The only branch in this code is the branch for deciding to stay in the loop vs exit the loop
  //this will diverge if two ants ready lists are of different sizes
  size_t maxIndx=0, fpIndx=0;
  pheromone_t max = -1, scoreProgress = 0;
  bool foundFPIndx = false;
  for (size_t i = 0; i < ready.size(); ++i) {
    const Choice &choice = ready[i];

    //code for picking the max
    bool CurrIsMax = choice.Score > max;
    max = CurrIsMax ? choice.Score : max;
    maxIndx = CurrIsMax ? i : maxIndx;

    //code for picking the fitness proportional choice
    scoreProgress += choice.Score;
    bool PastPoint = scoreProgress >= point;
    bool SetFP = PastPoint & !foundFPIndx;
    fpIndx = SetFP ? i : fpIndx;
    foundFPIndx |= PastPoint;
  }

  //finally we pick whether we will return the fp choice or max score inst w/o using a branch
  bool UseMax = rand < choose_best_chance;
  size_t indx = UseMax ? maxIndx : fpIndx;
  return ready[indx];

}

__host__ __device__
InstSchedule *ACOScheduler::FindOneSchedule(InstCount RPTarget, 
                                            InstSchedule *dev_schedule,
		                            DeviceVector<Choice> *dev_ready) {
#ifdef __CUDA_ARCH__ // device version of function
  SchedInstruction *inst = NULL;
  SchedInstruction *lastInst = NULL;
  InstSchedule *schedule = dev_schedule;
  InstCount maxPriority = dev_rdyLst_->MaxPriority();
  bool IsSecondPass = dev_rgn_->IsSecondPass();
  if (maxPriority == 0)
    maxPriority = 1; // divide by 0 is bad
  Initialize_();
  ((BBWithSpill *)dev_rgn_)->Dev_InitForSchdulng();

  SchedInstruction *waitFor = NULL;
  InstCount waitUntil = 0;
  double maxPriorityInv = 1 / maxPriority;
  DeviceVector<Choice> *ready = dev_ready;
  while (!IsSchedComplete_()) {
    UpdtRdyLst_(dev_crntCycleNum_[GLOBALTID], dev_crntSlotNum_[GLOBALTID]);

    // there are two steps to scheduling an instruction:
    // 1)Select the instruction(if we are not waiting on another instruction)
    if (!waitFor) {
      // if we have not already committed to schedule an instruction
      // next then pick one. First add ready instructions.  Including
      //"illegal" e.g. blocked instructions

      // convert the ready list from a custom priority queue to a std::vector,
      // much nicer for this particular scheduler
      double ScoreSum=0;
      unsigned long heuristic;
      ready->reserve(dev_rdyLst_->GetInstCnt());
      SchedInstruction *rInst = dev_rdyLst_->GetNextPriorityInst(heuristic);
      while (rInst != NULL) {
        if (ACO_SCHED_STALLS || ChkInstLglty_(inst)) {
          Choice c;
          c.inst = rInst;
          c.heuristic = (double)heuristic * maxPriorityInv + 1;
          c.readyOn = 0;
          c.Score = Score(lastInst,c);
          ScoreSum += c.Score;
          ready->push_back(c);
        }
        rInst = dev_rdyLst_->GetNextPriorityInst(heuristic);
      }
      dev_rdyLst_->ResetIterator();

#if ACO_SCHED_STALLS
      // add all instructions that are waiting due to latency to the choices
      // list
      PriorityArrayList<InstCount, InstCount> *lst = 
	                         dev_instsWithPrdcsrsSchduld_[GLOBALTID];
      SchedInstruction *fIns;
      for (InstCount fInstNum = lst->GetLastElmnt(); fInstNum != END;
           fInstNum = lst->GetPrevElmnt()) {
        fIns = dataDepGraph_->GetInstByIndx(fInstNum);
        bool changed;
        unsigned long heuristic = dev_rdyLst_->CmputKey_(fIns, false, changed);
        Choice c;
        c.inst = fIns;
        c.heuristic = (double)heuristic * maxPriorityInv + 1;
        c.readyOn = lst->GetCrntKey();
        c.Score = Score(lastInst,c);
        ScoreSum += c.Score;
        ready->push_back(c);
      }
      lst->ResetIterator();
#endif
      
      if (!ready->empty()) {
        Choice Sel = SelectInstruction(*ready, lastInst, ScoreSum);
        waitUntil = Sel.readyOn;
        inst = Sel.inst;
        if (waitUntil > dev_crntCycleNum_[GLOBALTID] || !ChkInstLglty_(inst)) {
          waitFor = inst;
          inst = NULL;
        }
      }
      if (inst != NULL) {
#if USE_ACS
        // local pheromone decay
        pheromone_t *pheromone = &Pheromone(lastInst, inst);
        *pheromone = 
          (1 - local_decay) * *pheromone + local_decay * initialValue_;
#endif
        lastInst = inst;
      }
    }

    // 2)Schedule a stall if we are still waiting, Schedule the instruction we
    // are waiting for if possible, decrement waiting time
    if (waitFor && waitUntil <= dev_crntCycleNum_[GLOBALTID]) {
      if (ChkInstLglty_(waitFor)) {
        inst = waitFor;
        waitFor = NULL;
      }
    }

    // boilerplate, mostly copied from ListScheduler, try not to touch it
    InstCount instNum;
    if (inst == NULL) {
      instNum = SCHD_STALL;
    } else {
      instNum = inst->GetNum();
      SchdulInst_(inst, dev_crntCycleNum_[GLOBALTID]);
      inst->Schedule(dev_crntCycleNum_[GLOBALTID],
                     dev_crntSlotNum_[GLOBALTID]);
      // In the second pass, calculate cost incrementally and terminate
      // ants that violate the RPTarget early
      ((BBWithSpill *)dev_rgn_)->Dev_SchdulInst(inst,
                                            dev_crntCycleNum_[GLOBALTID],
                                            dev_crntSlotNum_[GLOBALTID],
                                            false);
      // If an ant violates the RP cost constraint, terminate further
      // schedule construction
      if (((BBWithSpill *)dev_rgn_)->GetCrntSpillCost() > RPTarget) {
        // set schedule cost to INVALID_VALUE so it is not considered for
        // iteration best or global best
        schedule->SetCost(INVALID_VALUE);
        // keep track of ants terminated
        atomicAdd(&numAntsTerminated_, 1);
        dev_rdyLst_->ResetIterator();
        dev_rdyLst_->Reset();
        ready->clear();
        dev_instsWithPrdcsrsSchduld_[GLOBALTID]->Reset();
        // end schedule construction
        return NULL;
      } 
      DoRsrvSlots_(inst);
      // this is annoying
      SchedInstruction *blah = dev_rdyLst_->GetNextPriorityInst();
      while (blah != NULL && blah != inst) {
        blah = dev_rdyLst_->GetNextPriorityInst();
      }
      if (blah == inst)
        dev_rdyLst_->RemoveNextPriorityInst();
      UpdtSlotAvlblty_(inst);
    }
    schedule->AppendInst(instNum);
    if (MovToNxtSlot_(inst))
      InitNewCycle_();
    dev_rdyLst_->ResetIterator();
    ready->clear();
  }
  dev_rgn_->UpdateScheduleCost(schedule);
  return schedule;

#else  // **** Host version of function ****
  SchedInstruction *lastInst = NULL;
  InstSchedule *schedule;
  schedule = new InstSchedule(machMdl_, dataDepGraph_, true);
  InstCount maxPriority = rdyLst_->MaxPriority();
  bool IsSecondPass = rgn_->IsSecondPass();
  if (maxPriority == 0)
    maxPriority = 1; // divide by 0 is bad
  Initialize_();
  rgn_->InitForSchdulng();

  SchedInstruction *waitFor = NULL;
  InstCount waitUntil = 0;
  double maxPriorityInv = 1 / maxPriority;
  DeviceVector<Choice> *ready = 
      new DeviceVector<Choice>(dataDepGraph_->GetInstCnt());
  SchedInstruction *inst = NULL;
  while (!IsSchedComplete_()) {
    UpdtRdyLst_(crntCycleNum_, crntSlotNum_);

    // there are two steps to scheduling an instruction:
    // 1)Select the instruction(if we are not waiting on another instruction)
    inst = NULL;
    if (!waitFor) {
      // if we have not already committed to schedule an instruction
      // next then pick one. First add ready instructions.  Including
      //"illegal" e.g. blocked instructions

      // convert the ready list from a custom priority queue to a std::vector,
      // much nicer for this particular scheduler
      double ScoreSum=0;
      unsigned long heuristic;
      ready->reserve(rdyLst_->GetInstCnt());
      SchedInstruction *rInst = rdyLst_->GetNextPriorityInst(heuristic);
      while (rInst != NULL) {
        if (ACO_SCHED_STALLS || ChkInstLglty_(rInst)) {
          Choice c;
          c.inst = rInst;
          c.heuristic = (double)heuristic * maxPriorityInv + 1;
          c.readyOn = 0;
          c.Score = Score(lastInst,c);
          ScoreSum += c.Score;
          ready->push_back(c);
        }
        rInst = rdyLst_->GetNextPriorityInst(heuristic);
      }
      rdyLst_->ResetIterator();

#if ACO_SCHED_STALLS
      // add all instructions that are waiting due to latency to the choices
      // list
      for (InstCount fCycle = 1; fCycle < dataDepGraph_->GetMaxLtncy() &&
                                 crntCycleNum_ + fCycle < schedUprBound_;
           ++fCycle) {
        ArrayList<InstCount> *futureReady =
            frstRdyLstPerCycle_[crntCycleNum_ + fCycle];
        if (!futureReady)
          continue;

        SchedInstruction *fIns;
        for (InstCount fInstNum = futureReady->GetFrstElmnt(); fInstNum != END;
             fInstNum = futureReady->GetNxtElmnt()) {
          fIns = dataDepGraph_->GetInstByIndx(fInstNum);
          bool changed;
          unsigned long heuristic = rdyLst_->CmputKey_(fIns, false, changed);
          Choice c;
          c.inst = fIns;
          c.heuristic = (double)heuristic * maxPriorityInv + 1;
          c.readyOn = crntCycleNum_ + fCycle;
          c.Score = Score(lastInst,c);
          ScoreSum += c.Score;
          ready->push_back(c);
        }
        futureReady->ResetIterator();
      }
#endif

      if (!ready->empty()) {
        Choice Sel = SelectInstruction(*ready, lastInst, ScoreSum);
        waitUntil = Sel.readyOn;
        inst = Sel.inst;
        if (waitUntil > crntCycleNum_ || !ChkInstLglty_(inst)) {
          waitFor = inst;
          inst = NULL;
        }
      }
      if (inst != NULL) {
  #if USE_ACS
        // local pheromone decay
        pheromone_t *pheromone = &Pheromone(lastInst, inst);
        *pheromone = (1 - local_decay) * *pheromone + local_decay * initialValue_;
  #endif
        lastInst = inst;
      }
    }

    // 2)Schedule a stall if we are still waiting, Schedule the instruction we
    // are waiting for if possible, decrement waiting time
    if (waitFor && waitUntil <= crntCycleNum_) {
      if (ChkInstLglty_(waitFor)) {
        inst = waitFor;
        waitFor = NULL;
      }
    }

    // boilerplate, mostly copied from ListScheduler, try not to touch it
    InstCount instNum;
    if (inst == NULL) {
      instNum = SCHD_STALL;
    } else {
      instNum = inst->GetNum();
      SchdulInst_(inst, crntCycleNum_);
      inst->Schedule(crntCycleNum_, crntSlotNum_);
      rgn_->SchdulInst(inst, crntCycleNum_, crntSlotNum_, false);
      // If an ant violates the RP cost constraint, terminate further
      // schedule construction
      if (((BBWithSpill*)rgn_)->GetCrntSpillCost() > RPTarget) {
        // end schedule construction
        // keep track of ants terminated
        numAntsTerminated_++;
        delete rdyLst_;
        delete ready;
        rdyLst_ = new ReadyList(dataDepGraph_, prirts_); 
        delete schedule;
        return NULL;
      }
      DoRsrvSlots_(inst);
      // this is annoying
      SchedInstruction *blah = rdyLst_->GetNextPriorityInst();
      while (blah != NULL && blah != inst) {
        blah = rdyLst_->GetNextPriorityInst();
      }
      if (blah == inst)
        rdyLst_->RemoveNextPriorityInst();
      UpdtSlotAvlblty_(inst);
    }
    /* Logger::Info("Chose instruction %d (for some reason)", instNum); */
    schedule->AppendInst(instNum);
    if (MovToNxtSlot_(inst))
      InitNewCycle_();
    rdyLst_->ResetIterator();
    ready->clear();
  }
  delete ready;
  rgn_->UpdateScheduleCost(schedule);
  return schedule;
#endif
}

// Reduce to only index of best schedule per 2 blocks in output array
__inline__ __device__
void reduceToBestSchedPerBlock(InstSchedule **dev_schedules, int *blockBestIndex, ACOScheduler *dev_AcoSchdulr) {
  __shared__ int sdata[NUMTHREADSPERBLOCK];
  uint gtid = GLOBALTID;
  uint tid = threadIdx.x;
  int blockSize = NUMTHREADSPERBLOCK;
  
  // load candidate schedules into smem
  if (dev_AcoSchdulr->shouldReplaceSchedule(dev_schedules[gtid * 2], dev_schedules[gtid * 2 + 1], false))
    sdata[tid] = gtid * 2 + 1;
  else
    sdata[tid] = gtid * 2;
  __syncthreads();

  // do reduction on indexes in shared mem
  for (uint s = 1; s < blockDim.x; s*=2) {
    if (tid%(2*s) == 0) {
      if (dev_AcoSchdulr->shouldReplaceSchedule(
          dev_schedules[sdata[tid]], 
          dev_schedules[sdata[tid + s]], false))
        sdata[tid] = sdata[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0)
    blockBestIndex[blockIdx.x] = sdata[0];

}

// 1 block only to allow proper synchronization
// reduce to only one best index. At the end of this function globalBestIndex
// should be in blockBestIndex[0]
__inline__ __device__
void reduceToBestSched(InstSchedule **dev_schedules, int *blockBestIndex, ACOScheduler *dev_AcoSchdulr) {
  __shared__ int sBestIndex[NUMBLOCKS/4];
  uint tid = threadIdx.x;
  int index, sBestIndex1, sBestIndex2;
  
  // Load best indices into shared mem, reduce by half while doing so
  // If there are more than 64 schedules in blockBestIndex, some threads
  // will have to load in more than one value
  while (tid < NUMBLOCKS/4) {
    if (dev_AcoSchdulr->shouldReplaceSchedule(dev_schedules[blockBestIndex[tid * 2]], 
                                              dev_schedules[blockBestIndex[tid * 2 + 1]], false))
      sBestIndex[tid] = blockBestIndex[tid * 2 + 1];
    else
      sBestIndex[tid] = blockBestIndex[tid * 2];

    tid += blockDim.x;
  }
  __syncthreads();

  // reduce in smem
  for (uint s = 1; s < NUMBLOCKS/4; s *= 2) {
    tid = threadIdx.x;
    // if there are more than 32 schedules in smem, a thread
    // may reduce more than once per loop
    while (tid < NUMBLOCKS/4) {
      index = 2 * s * tid;

      if (index < NUMBLOCKS/4) {
        sBestIndex1 = sBestIndex[index];
        sBestIndex2 = sBestIndex[index + s];
        if (dev_AcoSchdulr->shouldReplaceSchedule(
            dev_schedules[sBestIndex1],
            dev_schedules[sBestIndex2], false))
          sBestIndex[index] = sBestIndex2;
      }
      tid += blockDim.x;
    }
    __syncthreads();
  }
  if (threadIdx.x == 0)
    blockBestIndex[0] = sBestIndex[0];
}

// default pheromone update scheme, one iteration best schedule is used to
// update pheromones every iteration
#define ONE_PER_ITER 0
// Update pheromones with best schedule found by each block every iteration
#define ONE_PER_BLOCK 1
// Update pheromones with all schedules
#define ALL 2
// select which pheromone update scheme to use
#define PHER_UPDATE_SCHEME ONE_PER_ITER

__device__ int globalBestIndex, dev_noImprovement;

__global__
void Dev_ACO(SchedRegion *dev_rgn, DataDepGraph *dev_DDG,
            ACOScheduler *dev_AcoSchdulr, InstSchedule **dev_schedules,
            DeviceVector<Choice> *dev_ready, InstSchedule *dev_bestSched,
            int noImprovementMax, int *blockBestIndex) {
  // holds cost and index of bestSched per block
  __shared__ int bestIndex, dev_iterations;
  __shared__ bool needsSLIL;
  needsSLIL = ((BBWithSpill *)dev_rgn)->needsSLIL();
  bool IsSecondPass = dev_rgn->IsSecondPass();
  dev_rgn->SetDepGraph(dev_DDG);
  ((BBWithSpill *)dev_rgn)->SetRegFiles(dev_DDG->getRegFiles());
  dev_noImprovement = 0;
  dev_iterations = 0;
  // Used to synchronize all launched threads
  auto threadGroup = cg::this_grid();
  // Get RPTarget
  InstCount RPTarget;

  // If in second pass and not using SLIL, set RPTarget
  if (!needsSLIL)
    RPTarget = dev_bestSched->GetSpillCost();
  else
    RPTarget = INT_MAX;

  // Start ACO
  while (dev_noImprovement < noImprovementMax) {
    // Reset schedules to post constructor state
    dev_schedules[GLOBALTID]->Initialize();
    dev_AcoSchdulr->FindOneSchedule(RPTarget,
                                    dev_schedules[GLOBALTID],
                                    &dev_ready[GLOBALTID]);
    // Sync threads after schedule creation
    threadGroup.sync();
    globalBestIndex = INVALID_VALUE;
    // reduce dev_schedules to 1 best schedule per block
    if (GLOBALTID < NUMTHREADS/2)
      reduceToBestSchedPerBlock(dev_schedules, blockBestIndex, dev_AcoSchdulr);

    threadGroup.sync();

    // one block to reduce blockBest schedules to one best schedule
    if (blockIdx.x == 0)
      reduceToBestSched(dev_schedules, blockBestIndex, dev_AcoSchdulr);

    threadGroup.sync();    

    if (GLOBALTID == 0 && 
        dev_schedules[blockBestIndex[0]]->GetCost() != INVALID_VALUE)
      globalBestIndex = blockBestIndex[0];

    // perform pheremone update based on selected scheme
#if (PHER_UPDATE_SCHEME == ONE_PER_ITER)
    // Another hard sync point after iteration best selection
    threadGroup.sync();
    if (globalBestIndex != INVALID_VALUE) 
      dev_AcoSchdulr->UpdatePheromone(dev_schedules[globalBestIndex]);
#elif (PHER_UPDATE_SCHEME == ONE_PER_BLOCK)
    // each block finds its blockIterationBest
    if (threadIdx.x == 0) {
      bestCost = dev_schedules[GLOBALTID]->GetCost();
      bestIndex = GLOBALTID; 
      for (int i = GLOBALTID + 1; i < GLOBALTID + NUMTHREADSPERBLOCK; i++) {
        if (dev_schedules[i]->GetCost() < bestCost) {
          bestCost = dev_schedules[i]->GetCost();
          bestIndex = i; 
        }
      }   
    }
    // wait for thread 0 of each block to find blockIterationBest
    threadGroup.sync();
    dev_AcoSchdulr->UpdatePheromone(dev_schedules[bestIndex]);
#elif (PHER_UPDATE_SCHEME == ALL)
    // each block loops over all schedules created by its threads and
    // updates pheromones in block level parallel
    for (int i = blockIdx.x * NUMTHREADSPERBLOCK; 
         i < ((blockIdx.x + 1) * NUMTHREADSPERBLOCK); i++) {
      dev_AcoSchdulr->UpdatePheromone(dev_schedules[i]);
    }
#endif
    // 1 thread compares iteration best to overall bestsched
    if (GLOBALTID == 0) {
      // Compare to initialSched/current best
      if (globalBestIndex != INVALID_VALUE &&
          dev_AcoSchdulr->shouldReplaceSchedule(dev_bestSched, 
                                                dev_schedules[globalBestIndex], 
                                                true)) {
        dev_bestSched->Copy(dev_schedules[globalBestIndex]);
        // update RPTarget if we are in second pass and not using SLIL
        if (!needsSLIL)
          RPTarget = dev_bestSched->GetSpillCost();
        printf("New best sched found by thread %d\n", globalBestIndex);
        printf("ACO found schedule "
               "cost:%d, rp cost:%d, exec cost: %d, and "
               "iteration:%d"
               " (sched length: %d, abs rp cost: %d, rplb: %d)\n",
             dev_bestSched->GetCost(), dev_bestSched->GetNormSpillCost(),
             dev_bestSched->GetExecCost(), dev_iterations,
             dev_bestSched->GetCrntLngth(), dev_bestSched->GetSpillCost(),
             dev_rgn->GetRPCostLwrBound());
#if !RUNTIME_TESTING
          dev_noImprovement = 0;
#else
          // for testing compile times disable resetting dev_noImprovement to
          // allow the same number of iterations every time
          atomicAdd(&dev_noImprovement, 1);
#endif     
      } else {
        atomicAdd(&dev_noImprovement, 1);
        if (dev_noImprovement > noImprovementMax)
          break;
      }
    }
    // wait for other blocks to finish before starting next iteration
    threadGroup.sync();
    // make sure no threads reset schedule before above operations complete
    if (threadIdx.x == 0)
      dev_iterations++;
  }
  if (GLOBALTID == 0) {
    printf("ACO finished after %d iterations\n", dev_iterations);
    printf("%d ants terminated early\n", dev_AcoSchdulr->GetNumAntsTerminated());
  }
}

FUNC_RESULT ACOScheduler::FindSchedule(InstSchedule *schedule_out,
                                       SchedRegion *region,
				       ACOScheduler *dev_AcoSchdulr) {
  rgn_ = region;

  // get settings
  Config &schedIni = SchedulerOptions::getInstance();
  bool IsFirst = !rgn_->IsSecondPass();
  heuristicImportance_ = schedIni.GetInt(
      IsFirst ? "ACO_HEURISTIC_IMPORTANCE" : "ACO2P_HEURISTIC_IMPORTANCE");
  if (dev_AcoSchdulr)
    dev_AcoSchdulr->heuristicImportance_ = heuristicImportance_;
  fixed_bias = schedIni.GetInt(IsFirst ? "ACO_FIXED_BIAS" : "ACO2P_FIXED_BIAS");
  if (dev_AcoSchdulr)
    dev_AcoSchdulr->fixed_bias = fixed_bias;
  noImprovementMax = schedIni.GetInt(IsFirst ? "ACO_STOP_ITERATIONS"
                                             : "ACO2P_STOP_ITERATIONS");
  if (dev_AcoSchdulr)
    dev_AcoSchdulr->noImprovementMax = noImprovementMax;

  // compute the relative maximum score inverse
  ScRelMax = rgn_->GetHeuristicCost();

  // initialize pheromone
  // for this, we need the cost of the pure heuristic schedule
  int pheromone_size = (count_ + 1) * count_;
  for (int i = 0; i < pheromone_size; i++)
    pheromone_[i] = 1;
  initialValue_ = 1;
  InstCount MaxRPTarget = std::numeric_limits<InstCount>::max();
  InstSchedule *heuristicSched = FindOneSchedule(MaxRPTarget);
  InstCount heuristicCost =
      heuristicSched->GetCost() + 1; // prevent divide by zero
  InstCount InitialCost = InitialSchedule ? InitialSchedule->GetCost() : 0;

#if USE_ACS
  initialValue_ = 2.0 / ((double)count_ * heuristicCost);
#else
  initialValue_ = (double)NUMTHREADS / heuristicCost;
#endif
  for (int i = 0; i < pheromone_size; i++)
    pheromone_[i] = initialValue_;
  std::cerr << "initialValue_" << initialValue_ << std::endl;
  InstSchedule *bestSchedule = InitialSchedule;
  if (bestSchedule) {
    UpdatePheromone(bestSchedule);
  }
  int noImprovement = 0; // how many iterations with no improvement
  int iterations = 0;
  InstSchedule *iterationBest = nullptr;

  if (DEV_ACO) { // Run ACO on device
    size_t memSize;
    // Update pheromones on device
    CopyPheromonesToDevice(dev_AcoSchdulr);
    Logger::Info("Creating and copying schedules to device"); 
    // An array to temporarily hold schedules to be copied over
    memSize = sizeof(InstSchedule) * NUMTHREADS;
    InstSchedule *temp_schedules = (InstSchedule *)malloc(memSize);
    // An array of pointers to schedules which are copied over
    InstSchedule **host_schedules = new InstSchedule *[NUMTHREADS];
    // Allocate one large array that will be split up between the dev arrays
    // of all InstSchedules. Massively decrease calls to hipMalloc/Free
    InstCount *dev_temp;
    size_t sizePerSched = bestSchedule->GetSizeOfDevArrays();
    memSize = sizePerSched * NUMTHREADS * sizeof(InstCount);
    gpuErrchk(hipMalloc(&dev_temp, memSize));
    memSize = sizeof(InstSchedule);
    for (int i = 0; i < NUMTHREADS; i++) {
      // Create new schedule
      host_schedules[i] = new InstSchedule(machMdl_, dataDepGraph_, true);
      // Pass a dev array to the schedule to be divided up between the required
      // dev arrays for InstSchedule
      host_schedules[i]->SetDevArrayPointers(dev_MM_, 
                                             &dev_temp[i*sizePerSched]);
      // Copy to temp_schedules array to later copy to device with 1 hipMemcpy
      memcpy(&temp_schedules[i], host_schedules[i], memSize);
    }
    // Allocate and Copy array of schedules to device
    // A device array of schedules
    InstSchedule *dev_schedules_arr;
    memSize = sizeof(InstSchedule) * NUMTHREADS;
    gpuErrchk(hipMalloc(&dev_schedules_arr, memSize));
    // Copy schedules to device
    gpuErrchk(hipMemcpy(dev_schedules_arr, temp_schedules, memSize,
                         hipMemcpyHostToDevice));
    free(temp_schedules);
    // Create a dev array of pointers to dev_schedules_arr
    // Passing and array of schedules and dereferencing the array
    // to get pointers slows down the kernel significantly
    InstSchedule **dev_schedules;
    memSize = sizeof(InstSchedule *) * NUMTHREADS;
    gpuErrchk(hipMallocManaged(&dev_schedules, memSize));
    for (int i = 0; i < NUMTHREADS; i++)
      dev_schedules[i] = &dev_schedules_arr[i];
    gpuErrchk(hipMemPrefetchAsync(dev_schedules, memSize, 0));
    // Copy over best schedule
    // holds device copy of best sched, to be copied back to host after kernel
    InstSchedule *dev_bestSched;
    bestSchedule = new InstSchedule(machMdl_, dataDepGraph_, true);
    bestSchedule->Copy(InitialSchedule);
    bestSchedule->AllocateOnDevice(dev_MM_);
    bestSchedule->CopyArraysToDevice();
    memSize = sizeof(InstSchedule);
    gpuErrchk(hipMalloc((void**)&dev_bestSched, memSize));
    gpuErrchk(hipMemcpy(dev_bestSched, bestSchedule, memSize,
                         hipMemcpyHostToDevice));
    // Create a global mem array for device to use in parallel reduction
    int *dev_blockBestIndex;
    memSize = (NUMBLOCKS/2) * sizeof(int);
    gpuErrchk(hipMalloc(&dev_blockBestIndex, memSize));
    // Make sure managed memory is copied to device before kernel start
    memSize = sizeof(ACOScheduler);
    gpuErrchk(hipMemPrefetchAsync(dev_AcoSchdulr, memSize, 0));
    Logger::Info("Launching Dev_ACO with %d blocks of %d threads", NUMBLOCKS,
                                                           NUMTHREADSPERBLOCK);
    // Using Cooperative Grid Groups requires launching with
    // hipLaunchCooperativeKernel which requires kernel args to be an array
    // of void pointers to host memory locations of the arguments
    dim3 gridDim(NUMBLOCKS);
    dim3 blockDim(NUMTHREADSPERBLOCK);
    void *dArgs[8];
    dArgs[0] = (void*)&dev_rgn_;
    dArgs[1] = (void*)&dev_DDG_;
    dArgs[2] = (void*)&dev_AcoSchdulr;
    dArgs[3] = (void*)&dev_schedules;
    dArgs[4] = (void*)&dev_ready_;
    dArgs[5] = (void*)&dev_bestSched;
    dArgs[6] = (void*)&noImprovementMax;
    dArgs[7] = (void*)&dev_blockBestIndex;
    gpuErrchk(hipLaunchCooperativeKernel((void*)Dev_ACO, gridDim, blockDim, 
                                          dArgs));
    hipDeviceSynchronize();
    Logger::Info("Post Kernel Error: %s", 
                 hipGetErrorString(hipGetLastError()));
    // Copy dev_bestSched back to host
    memSize = sizeof(InstSchedule);
    gpuErrchk(hipMemcpy(bestSchedule, dev_bestSched, memSize,
                         hipMemcpyDeviceToHost));
    bestSchedule->CopyArraysToHost();
    // Free allocated memory that is no longer needed
    bestSchedule->FreeDeviceArrays();
    hipFree(dev_bestSched);
    for (int i = 0; i < NUMTHREADS; i++) {
      delete host_schedules[i];
    }
    delete[] host_schedules;
    // delete the large array shared by all schedules
    hipFree(dev_temp);
    hipFree(dev_schedules);

  } else { // Run ACO on cpu
    Logger::Info("Running host ACO with %d ants per iteration", NUMTHREADS);
    InstCount RPTarget;
    if (!((BBWithSpill *)rgn_)->needsSLIL())
      RPTarget = bestSchedule->GetSpillCost();
    else
      RPTarget = MaxRPTarget;
    while (noImprovement < noImprovementMax) {
      iterationBest = nullptr;
      for (int i = 0; i < NUMTHREADS; i++) {
        InstSchedule *schedule = FindOneSchedule(RPTarget);
        if (print_aco_trace)
          PrintSchedule(schedule);
        if (shouldReplaceSchedule(iterationBest, schedule, false)) {
          if (iterationBest)
            delete iterationBest;          
          iterationBest = schedule;
        } else {
            if (schedule)
              delete schedule;
        }
      }
#if !USE_ACS
      if (iterationBest)
        UpdatePheromone(iterationBest);
#endif
      if (shouldReplaceSchedule(bestSchedule, iterationBest, true)) {
        if (bestSchedule && bestSchedule != InitialSchedule)
          delete bestSchedule;
        bestSchedule = std::move(iterationBest);
        if (!((BBWithSpill *)rgn_)->needsSLIL())
          RPTarget = bestSchedule->GetSpillCost();
        printf("ACO found schedule "
               "cost:%d, rp cost:%d, sched length: %d, and "
               "iteration:%d\n",
               bestSchedule->GetCost(), bestSchedule->GetSpillCost(),
               bestSchedule->GetCrntLngth(), iterations);
#if !RUNTIME_TESTING
          noImprovement = 0;
#else
          // Disable resetting noImp to lock iterations to 10
          noImprovement++;
#endif
      } else {
        delete iterationBest;
        noImprovement++;
      }
#if USE_ACS
      UpdatePheromone(bestSchedule);
#endif
      iterations++;
    }
    Logger::Info("%d ants terminated early", numAntsTerminated_);
  } // End run on CPU

  printf("Best schedule: ");
  PrintSchedule(bestSchedule);
  schedule_out->Copy(bestSchedule);
  if (bestSchedule != InitialSchedule)
    delete bestSchedule;
  if (!DEV_ACO)
    printf("ACO finished after %d iterations\n", iterations);

  return RES_SUCCESS;
}

__host__ __device__
void ACOScheduler::UpdatePheromone(InstSchedule *schedule) {
#ifdef __CUDA_ARCH__ // device version of function
#if (PHER_UPDATE_SCHEME == ONE_PER_ITER)
  // parallel on global level
  int instNum = GLOBALTID;
#elif (PHER_UPDATE_SCHEME == ALL || PHER_UPDATE_SCHEME == ONE_PER_BLOCK)
  // parallel on block level
  int instNum = threadIdx.x;
#endif
  // Each thread updates pheromone table for 1 instruction
  // For the case NUMTHREADS < count_, increase instNum by 
  // NUMTHREADS at the end of the loop.
  InstCount lastInstNum = -1;
  thrust::maximum<double> dmax;
  thrust::minimum<double> dmin;
  pheromone_t portion = schedule->GetCost() / (ScRelMax * 1.5);
  pheromone_t deposition;
  if (portion < 1)
    deposition = (1 - portion) * MAX_DEPOSITION_MINUS_MIN + MIN_DEPOSITION;
  else
    deposition = MIN_DEPOSITION;

  pheromone_t *pheromone;
  while (instNum < count_) {
    // Get the instruction that comes before inst in the schedule
    // if instNum == count_ - 2 it has the root inst and lastInstNum = -1
    lastInstNum = schedule->GetPrevInstNum(instNum);
    // Get corresponding pheromone and update it
    pheromone = &Pheromone(lastInstNum, instNum);
    *pheromone = *pheromone + deposition;
    // decay pheromone for all trails leading to instNum
    for (int j = 0; j < count_; j++) {
      pheromone = &Pheromone(j, instNum);
      *pheromone *= (1 - decay_factor);
      *pheromone = dmax(1, dmin(8, *pheromone));
    }
#if (PHER_UPDATE_SCHEME == ONE_PER_ITER)
    // parallel on global level
    // Increase instNum by NUMTHREADS until over count_
    instNum += NUMTHREADS;
#elif (PHER_UPDATE_SCHEME == ALL || PHER_UPDATE_SCHEME == ONE_PER_BLOCK)
    // parallel on block level
    instNum += NUMTHREADSPERBLOCK;
#endif
  }
  if (print_aco_trace)
    PrintPheromone();

#else // host version of function
  // I wish InstSchedule allowed you to just iterate over it, but it's got this
  // cycle and slot thing which needs to be accounted for
  InstCount instNum, cycleNum, slotNum;
  instNum = schedule->GetFrstInst(cycleNum, slotNum);

  SchedInstruction *lastInst = NULL;
  pheromone_t portion = schedule->GetCost() / (ScRelMax * 1.5);
  pheromone_t deposition =
      fmax((1 - portion) * MAX_DEPOSITION_MINUS_MIN, 0) + MIN_DEPOSITION;
  pheromone_t *pheromone;
  while (instNum != INVALID_VALUE) {  
    SchedInstruction *inst = dataDepGraph_->GetInstByIndx(instNum);

    pheromone = &Pheromone(lastInst, inst);
#if USE_ACS
    // ACS update rule includes decay
    // only the arcs on the current solution are decayed
    *pheromone = (1 - decay_factor) * *pheromone +
                 decay_factor / (schedule->GetCost() + 1);
#else
    *pheromone = *pheromone + deposition;
#endif
    lastInst = inst;

    instNum = schedule->GetNxtInst(cycleNum, slotNum);
  }
  schedule->ResetInstIter();

#if !USE_ACS
  // decay pheromone
  for (int i = 0; i < count_; i++) {
    for (int j = 0; j < count_; j++) {
      pheromone = &Pheromone(j, instNum);
      *pheromone *= (1 - decay_factor);
      *pheromone = fmax(1, fmin(8, *pheromone));
    }
  }
#endif
  if (print_aco_trace)
    PrintPheromone();
#endif
}

__device__
void ACOScheduler::CopyPheromonesToSharedMem(double *s_pheromone) {
  InstCount toInstNum = threadIdx.x;
  while (toInstNum < count_) {
    for (int fromInstNum = -1; fromInstNum < count_; fromInstNum++)
      s_pheromone[((fromInstNum + 1) * count_) + toInstNum] = 
                                        Pheromone(fromInstNum, toInstNum);
    toInstNum += NUMTHREADSPERBLOCK;
  }
}

inline void ACOScheduler::UpdtRdyLst_(InstCount cycleNum, int slotNum) {
#ifdef __CUDA_ARCH__ // Device version
  InstCount prevCycleNum = cycleNum - 1;
  int lstSize = dev_instsWithPrdcsrsSchduld_[GLOBALTID]->size_;
  PriorityArrayList<InstCount, InstCount> *lst = 
	                         dev_instsWithPrdcsrsSchduld_[GLOBALTID];
  SchedInstruction *inst;
  // PriorityArrayList holds keys in decreasing order, so insts with earliest
  // rdyCycle are last on the list
  while (lstSize > 0 && lst->keys_[lstSize - 1] <= cycleNum) {
    inst = dataDepGraph_->GetInstByIndx(lst->elmnts_[lstSize - 1]);
    dev_rdyLst_->AddInst(inst);
    lst->RmvLastElmnt();
    lstSize--;
  }

#else  // Host version
  InstCount prevCycleNum = cycleNum - 1;
  ArrayList<InstCount> *lst1 = NULL;
  ArrayList<InstCount> *lst2 = frstRdyLstPerCycle_[cycleNum];

  if (slotNum == 0 && prevCycleNum >= 0) {
    // If at the begining of a new cycle other than the very first cycle, then
    // we also have to include the instructions that might have become ready in
    // the previous cycle due to a zero latency of the instruction scheduled in
    // the very last slot of that cycle [GOS 9.8.02].
    lst1 = frstRdyLstPerCycle_[prevCycleNum];

    if (lst1 != NULL) {
      rdyLst_->AddList(lst1);
      lst1->Reset();
      CleanupCycle_(prevCycleNum);
    }
  }

  if (lst2 != NULL) {
    rdyLst_->AddList(lst2);
    lst2->Reset();
  }
#endif
}

__host__ __device__
void ACOScheduler::PrintPheromone() {
  for (int i = 0; i < count_; i++) {
    for (int j = 0; j < count_; j++) {
      //std::cerr << std::scientific << std::setprecision(8) << Pheromone(i, j)
      //          << " ";
      printf("%.10e ", Pheromone(i, j));
    }
    //std::cerr << std::endl;
    printf("\n");
  }
  //std::cerr << std::endl;
  printf("\n");
}

#ifndef NDEBUG
// NOLINTNEXTLINE(clang-diagnostic-unused-function)
static void PrintInstruction(SchedInstruction *inst) {
  std::cerr << std::setw(2) << inst->GetNum() << " ";
  std::cerr << std::setw(20) << std::left << inst->GetOpCode();

  std::cerr << " defs ";
  Register **defs;
  uint16_t defsCount = inst->GetDefs(defs);
  for (uint16_t i = 0; i < defsCount; i++) {
    std::cerr << defs[i]->GetNum() << defs[i]->GetType();
    if (i != defsCount - 1)
      std::cerr << ", ";
  }

  std::cerr << " uses ";
  Register **uses;
  uint16_t usesCount = inst->GetUses(uses);
  for (uint16_t i = 0; i < usesCount; i++) {
    std::cerr << uses[i]->GetNum() << uses[i]->GetType();
    if (i != usesCount - 1)
      std::cerr << ", ";
  }
  std::cerr << std::endl;
}
#endif

__host__ __device__
void PrintSchedule(InstSchedule *schedule) {
  printf("%d: ", schedule->GetCost());
  InstCount instNum, cycleNum, slotNum;
  instNum = schedule->GetFrstInst(cycleNum, slotNum);
  while (instNum != INVALID_VALUE) {
    printf("%d ", instNum);
    instNum = schedule->GetNxtInst(cycleNum, slotNum);
  }
  printf("\n");
  schedule->ResetInstIter();
}

void ACOScheduler::setInitialSched(InstSchedule *Sched) {
  if (Sched) {
    InitialSchedule =
        new InstSchedule(machMdl_, dataDepGraph_, VrfySched_);
    InitialSchedule->Copy(Sched);
  }
}

void ACOScheduler::AllocDevArraysForParallelACO() {
  size_t memSize;
  // Alloc dev array for schduldInstCnt_
  memSize = sizeof(InstCount) * NUMTHREADS;
  gpuErrchk(hipMalloc(&dev_schduldInstCnt_, memSize));
  // Alloc dev array for crntCycleNum_;
  memSize = sizeof(InstCount) * NUMTHREADS;
  gpuErrchk(hipMalloc(&dev_crntCycleNum_, memSize));
  // Alloc dev array for crntSlotNum_;
  memSize = sizeof(InstCount) * NUMTHREADS;
  gpuErrchk(hipMalloc(&dev_crntSlotNum_, memSize));
  // Allo dev array for crntRealSlotNum_
  memSize = sizeof(InstCount) * NUMTHREADS;
  gpuErrchk(hipMalloc(&dev_crntRealSlotNum_, memSize));
  // Alloc dev array for isCrntCycleBlkd_;
  memSize = sizeof(bool) * NUMTHREADS;
  gpuErrchk(hipMalloc(&dev_isCrntCycleBlkd_, memSize));
  // Alloc dev array for rdyLst_
  rdyLst_->AllocDevArraysForParallelACO(NUMTHREADS);
  // Alloc dev array for avlblSlotsInCrntCycle_
  memSize = sizeof(int16_t *) * NUMTHREADS;
  gpuErrchk(hipMallocManaged(&dev_avlblSlotsInCrntCycle_, memSize));
  // Alloc dev arrays of avlblSlotsInCrntCycle_ for each thread
  memSize = sizeof(int16_t) * issuTypeCnt_;
  for (int i = 0; i < NUMTHREADS; i++) {
    gpuErrchk(hipMalloc(&dev_avlblSlotsInCrntCycle_[i], memSize));
  }
  // Alloc dev arrays for dev_instsWithPrdcsrsSchduld_
  memSize = sizeof(PriorityArrayList<InstCount, InstCount> *) * NUMTHREADS;
  gpuErrchk(hipMallocManaged(&dev_instsWithPrdcsrsSchduld_, memSize));
  // Alloc dev arrays for rsrvSlots_
  memSize = sizeof(ReserveSlot *) * NUMTHREADS;
  gpuErrchk(hipMallocManaged(&dev_rsrvSlots_, memSize));
  memSize = sizeof(ReserveSlot) * issuRate_;
  for (int i = 0; i < NUMTHREADS; i++) {
    gpuErrchk(hipMalloc(&dev_rsrvSlots_[i], memSize));
  }
  memSize = sizeof(int16_t) * NUMTHREADS;
  gpuErrchk(hipMalloc(&dev_rsrvSlotCnt_, memSize));
}

void ACOScheduler::CopyPheromonesToDevice(ACOScheduler *dev_AcoSchdulr) {
  size_t memSize;
  // Free allocated mem sinve pheromone size can change
  if (dev_AcoSchdulr->dev_pheromone_elmnts_alloced_ == true)
    hipFree(dev_AcoSchdulr->pheromone_.elmnts_);

  memSize = sizeof(DeviceVector<pheromone_t>);
  gpuErrchk(hipMemcpy(&dev_AcoSchdulr->pheromone_, &pheromone_, memSize,
            hipMemcpyHostToDevice));

  memSize = sizeof(pheromone_t) * pheromone_.alloc_;
  gpuErrchk(hipMalloc(&(dev_AcoSchdulr->pheromone_.elmnts_), memSize));
  gpuErrchk(hipMemcpy(dev_AcoSchdulr->pheromone_.elmnts_, pheromone_.elmnts_,
		       memSize, hipMemcpyHostToDevice));
  
  dev_AcoSchdulr->dev_pheromone_elmnts_alloced_ = true;
}

void ACOScheduler::CopyPointersToDevice(ACOScheduler *dev_ACOSchedulr) {
  size_t memSize;
  dev_ACOSchedulr->machMdl_ = dev_MM_;
  dev_ACOSchedulr->dataDepGraph_ = dev_DDG_;
  // Copy slotsPerTypePerCycle_
  int *dev_slotsPerTypePerCycle;
  memSize = sizeof(int) * issuTypeCnt_;
  gpuErrchk(hipMalloc(&dev_slotsPerTypePerCycle, memSize));
  gpuErrchk(hipMemcpy(dev_slotsPerTypePerCycle, slotsPerTypePerCycle_,
		       memSize, hipMemcpyHostToDevice));
  dev_ACOSchedulr->slotsPerTypePerCycle_ = dev_slotsPerTypePerCycle;
  // Copy instCntPerIssuType_
  InstCount *dev_instCntPerIssuType;
  memSize = sizeof(InstCount) * issuTypeCnt_;
  gpuErrchk(hipMalloc(&dev_instCntPerIssuType, memSize));
  gpuErrchk(hipMemcpy(dev_instCntPerIssuType, instCntPerIssuType_, memSize,
		       hipMemcpyHostToDevice));
  dev_ACOSchedulr->instCntPerIssuType_ = dev_instCntPerIssuType;
  // set root/leaf inst
  dev_ACOSchedulr->rootInst_ = dev_DDG_->GetRootInst();
  dev_ACOSchedulr->leafInst_ = dev_DDG_->GetLeafInst();
  // Create an array of PriorityArrayLists, allocate dev mem for it 
  // and elmnts_ for each one, and copy it to device
  PriorityArrayList<InstCount, InstCount> *temp = 
    new PriorityArrayList<InstCount, InstCount>[NUMTHREADS];
  // Allocate elmnts_ and keys_ for all PArrayLists
  InstCount *dev_elmnts, *dev_keys;
  memSize = sizeof(InstCount) * count_ * NUMTHREADS;
  gpuErrchk(hipMalloc(&dev_elmnts, memSize));
  gpuErrchk(hipMalloc(&dev_keys, memSize));
  // set correct maxSize, elmnts_, and keys_ for each PArrayList
  for (int i = 0; i < NUMTHREADS; i++) {
    temp[i].maxSize_ = count_;
    temp[i].elmnts_ = &dev_elmnts[i * count_];
    temp[i].keys_ = &dev_keys[i * count_];
  }
  // Alloc dev mem and copy array of PArrayLists to device
  PriorityArrayList<InstCount, InstCount> *dev_array;
  memSize = sizeof(PriorityArrayList<InstCount, InstCount>) * NUMTHREADS;
  gpuErrchk(hipMallocManaged(&dev_array, memSize));
  gpuErrchk(hipMemcpy(dev_array, temp, memSize, hipMemcpyHostToDevice));
  // set dev_instsWithPrdcsrsScheduld_ pointers to each PAL in array
  for (int i = 0; i < NUMTHREADS; i++)
    dev_instsWithPrdcsrsSchduld_[i] = &dev_array[i];
  // make sure host also has a copy of array for later deletion
  memSize = sizeof(PriorityArrayList<InstCount, InstCount>) * NUMTHREADS;
  gpuErrchk(hipMemPrefetchAsync(dev_array, memSize, hipCpuDeviceId));
  // remove references to dev arrays in host copy and delete host copy
  for (int i = 0; i < NUMTHREADS; i++) {
    temp[i].elmnts_ = NULL;
    temp[i].keys_ = NULL;
  }
  delete[] temp;

  // Copy rdyLst_
  memSize = sizeof(ReadyList);
  gpuErrchk(hipMallocManaged(&dev_ACOSchedulr->dev_rdyLst_, memSize));
  gpuErrchk(hipMemcpy(dev_ACOSchedulr->dev_rdyLst_, rdyLst_, memSize,
		       hipMemcpyHostToDevice));
  rdyLst_->CopyPointersToDevice(dev_ACOSchedulr->dev_rdyLst_, dev_DDG_,
		                NUMTHREADS);
  // make sure hipMallocManaged memory is copied to device before kernel start
  memSize = sizeof(int16_t *) * NUMTHREADS;
  gpuErrchk(hipMemPrefetchAsync(dev_avlblSlotsInCrntCycle_, memSize, 0));
  memSize = sizeof(PriorityArrayList<InstCount, InstCount> *) * NUMTHREADS;
  gpuErrchk(hipMemPrefetchAsync(dev_instsWithPrdcsrsSchduld_, memSize, 0));
  memSize = sizeof(ReserveSlot *) * NUMTHREADS;
  gpuErrchk(hipMemPrefetchAsync(dev_rsrvSlots_, memSize, 0));
  memSize = sizeof(ReadyList);
  gpuErrchk(hipMemPrefetchAsync(&dev_ACOSchedulr->dev_rdyLst_, memSize, 0));
}

void ACOScheduler::FreeDevicePointers() {
  hipFree(dev_schduldInstCnt_);
  hipFree(dev_crntCycleNum_);
  hipFree(dev_crntSlotNum_);
  hipFree(dev_crntRealSlotNum_);
  hipFree(dev_isCrntCycleBlkd_);
  hipFree(slotsPerTypePerCycle_);
  hipFree(instCntPerIssuType_);
  for (int i = 0; i < NUMTHREADS; i++){
    //hipFree(dev_instsWithPrdcsrsSchduld_[i]->elmnts_);
    //hipFree(dev_instsWithPrdcsrsSchduld_[i]->keys_);
    //hipFree(dev_instsWithPrdcsrsSchduld_[i]);
    hipFree(dev_avlblSlotsInCrntCycle_[i]);
    hipFree(dev_rsrvSlots_[i]);
  }
  hipFree(dev_instsWithPrdcsrsSchduld_[0]->elmnts_);
  hipFree(dev_instsWithPrdcsrsSchduld_[0]->keys_);
  hipFree(dev_instsWithPrdcsrsSchduld_[0]);
  dev_rdyLst_->FreeDevicePointers(NUMTHREADS);
  hipFree(dev_avlblSlotsInCrntCycle_);
  hipFree(dev_rsrvSlots_);
  hipFree(dev_rsrvSlotCnt_);
  hipFree(dev_instsWithPrdcsrsSchduld_);
  hipFree(dev_rdyLst_);
  hipFree(pheromone_.elmnts_);
}
