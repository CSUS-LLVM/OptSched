#include "opt-sched/Scheduler/ready_list.h"
#include "opt-sched/Scheduler/data_dep.h"
#include "opt-sched/Scheduler/logger.h"
#include "opt-sched/Scheduler/utilities.h"
#include "opt-sched/Scheduler/cuda_lnkd_lst.cuh"
#include "opt-sched/Scheduler/dev_defines.h"

using namespace llvm::opt_sched;

__host__ __device__
ReadyList::ReadyList(DataDepGraph *dataDepGraph, SchedPriorities prirts) {
  dataDepGraph_ = dataDepGraph;
  prirts_ = prirts;
  prirtyLst_ = NULL;
  int i;
  uint16_t totKeyBits = 0;

  // Initialize an array of KeyedEntry if a dynamic heuristic is used. This
  // enable fast updating for dynamic heuristics.
/*  if (prirts_.isDynmc)
    keyedEntries_ = new KeyedEntry<SchedInstruction, unsigned long>
        *[dataDepGraph->GetInstCnt()];
  else*/
    keyedEntries_ = nullptr;

  useCntBits_ = crtclPathBits_ = scsrCntBits_ = ltncySumBits_ = nodeID_Bits_ =
      inptSchedOrderBits_ = 0;

  // Calculate the number of bits needed to hold the maximum value of each
  // priority scheme
  for (i = 0; i < prirts.cnt; i++) {
    switch (prirts.vctr[i]) {
    case LSH_CP:
    case LSH_CPR:
      maxCrtclPath_ = dataDepGraph->GetRootInst()->GetCrntLwrBound(DIR_BKWRD);
      crtclPathBits_ = Utilities::clcltBitsNeededToHoldNum(maxCrtclPath_);
      totKeyBits += crtclPathBits_;
      break;

    case LSH_LUC:
      /*for (int j = 0; j < dataDepGraph->GetInstCnt(); j++) {
        keyedEntries_[j] = NULL;
      }*/
      maxUseCnt_ = dataDepGraph->GetMaxUseCnt();
      useCntBits_ = Utilities::clcltBitsNeededToHoldNum(maxUseCnt_);
      totKeyBits += useCntBits_;
      break;

    case LSH_UC:
      maxUseCnt_ = dataDepGraph->GetMaxUseCnt();
      useCntBits_ = Utilities::clcltBitsNeededToHoldNum(maxUseCnt_);
      totKeyBits += useCntBits_;
      break;

    case LSH_NID:
    case LSH_LLVM:
      maxNodeID_ = dataDepGraph->GetInstCnt() - 1;
      nodeID_Bits_ = Utilities::clcltBitsNeededToHoldNum(maxNodeID_);
      totKeyBits += nodeID_Bits_;
      break;

    case LSH_ISO:
      maxInptSchedOrder_ = dataDepGraph->GetMaxFileSchedOrder();
      inptSchedOrderBits_ =
          Utilities::clcltBitsNeededToHoldNum(maxInptSchedOrder_);
      totKeyBits += inptSchedOrderBits_;
      break;

    case LSH_SC:
      maxScsrCnt_ = dataDepGraph->GetMaxScsrCnt();
      scsrCntBits_ = Utilities::clcltBitsNeededToHoldNum(maxScsrCnt_);
      totKeyBits += scsrCntBits_;
      break;

    case LSH_LS:
      maxLtncySum_ = dataDepGraph->GetMaxLtncySum();
      ltncySumBits_ = Utilities::clcltBitsNeededToHoldNum(maxLtncySum_);
      totKeyBits += ltncySumBits_;
      break;
    } // end switch
  }   // end for

  assert(totKeyBits <= 8 * sizeof(unsigned long));

#ifdef IS_DEBUG_READY_LIST2
  Logger::Info("The ready list key size is %d bits", totKeyBits);
#endif

  prirtyLst_ = 
      new PriorityArrayList<InstCount>(dataDepGraph_->GetInstCnt());
  latestSubLst_ = new ArrayList<InstCount>(dataDepGraph_->GetInstCnt());

  int16_t keySize = 0;
  maxPriority_ = 0;
  for (i = 0; i < prirts_.cnt; i++) {
    switch (prirts_.vctr[i]) {
    case LSH_CP:
    case LSH_CPR:
      AddPrirtyToKey_(maxPriority_, keySize, crtclPathBits_, maxCrtclPath_,
                      maxCrtclPath_);
      break;
    case LSH_LUC:
    case LSH_UC:
      AddPrirtyToKey_(maxPriority_, keySize, useCntBits_, maxUseCnt_,
                      maxUseCnt_);
      break;
    case LSH_NID:
    case LSH_LLVM:
      AddPrirtyToKey_(maxPriority_, keySize, nodeID_Bits_, maxNodeID_,
                      maxNodeID_);
      break;
    case LSH_ISO:
      AddPrirtyToKey_(maxPriority_, keySize, inptSchedOrderBits_,
                      maxInptSchedOrder_, maxInptSchedOrder_);
      break;
    case LSH_SC:
      AddPrirtyToKey_(maxPriority_, keySize, scsrCntBits_, maxScsrCnt_,
                      maxScsrCnt_);
      break;
    case LSH_LS:
      AddPrirtyToKey_(maxPriority_, keySize, ltncySumBits_, maxLtncySum_,
                      maxLtncySum_);
      break;
    }
  }
}

__host__ __device__
ReadyList::~ReadyList() {
  Reset();
  if (prirtyLst_)
    delete prirtyLst_;
  if (latestSubLst_)
    delete latestSubLst_;
  if (keyedEntries_)
    delete keyedEntries_;
}

__host__ __device__
void ReadyList::Reset() {
#ifdef __CUDA_ARCH__
  dev_prirtyLst_[GLOBALTID].Reset();
#else
  prirtyLst_->Reset();
  latestSubLst_->Reset();
#endif
}

__host__ __device__
void ReadyList::CopyList(ReadyList *othrLst) {
  assert(prirtyLst_->GetElmntCnt() == 0);
  assert(latestSubLst_->GetElmntCnt() == 0);
  assert(othrLst != NULL);

  // Copy the ready list and create the array of keyed entries. If a dynamic
  // heuristic is not used then the second parameter should be a nullptr and the
  // array will not be created.
  prirtyLst_->CopyList(othrLst->prirtyLst_);
}

__device__ __host__
unsigned long ReadyList::CmputKey_(SchedInstruction *inst, bool isUpdate,
                                   bool &changed) {
  unsigned long key = 0;
  int16_t keySize = 0;
  int i;
  int16_t oldLastUseCnt, newLastUseCnt;
  changed = true;
  if (isUpdate)
    changed = false;

  for (i = 0; i < prirts_.cnt; i++) {
    switch (prirts_.vctr[i]) {
    case LSH_CP:
    case LSH_CPR:
      AddPrirtyToKey_(key, keySize, crtclPathBits_,
                      inst->GetCrtclPath(DIR_BKWRD), maxCrtclPath_);
      break;

    case LSH_LUC:
      oldLastUseCnt = inst->GetLastUseCnt();
      newLastUseCnt = inst->CmputLastUseCnt();
      if (newLastUseCnt != oldLastUseCnt)
        changed = true;

      AddPrirtyToKey_(key, keySize, useCntBits_, newLastUseCnt, maxUseCnt_);
      break;

    case LSH_UC:
      AddPrirtyToKey_(key, keySize, useCntBits_, inst->GetUseCnt(), maxUseCnt_);
      break;

    case LSH_NID:
    case LSH_LLVM:
      AddPrirtyToKey_(key, keySize, nodeID_Bits_,
                      maxNodeID_ - inst->GetNodeID(), maxNodeID_);
      break;

    case LSH_ISO:
      AddPrirtyToKey_(key, keySize, inptSchedOrderBits_,
                      maxInptSchedOrder_ - inst->GetFileSchedOrder(),
                      maxInptSchedOrder_);
      break;

    case LSH_SC:
      AddPrirtyToKey_(key, keySize, scsrCntBits_, inst->GetScsrCnt(),
                      maxScsrCnt_);
      break;

    case LSH_LS:
      AddPrirtyToKey_(key, keySize, ltncySumBits_, inst->GetLtncySum(),
                      maxLtncySum_);
      break;
    }
  }
  return key;
}

__host__ __device__
void ReadyList::AddLatestSubLists(ArrayList<InstCount> *lst1,
                                  ArrayList<InstCount> *lst2) {
  assert(latestSubLst_->GetElmntCnt() == 0);
  if (lst1 != NULL)
    AddLatestSubList_(lst1);
  if (lst2 != NULL)
    AddLatestSubList_(lst2);
  prirtyLst_->ResetIterator();
}

void ReadyList::Print(std::ostream &out) {
  out << "Ready List: ";
  for (auto crntInst = prirtyLst_->GetFrstElmnt(); crntInst != END;
       crntInst = prirtyLst_->GetNxtElmnt()) {
    out << " " << crntInst;
  }
  out << '\n';

  prirtyLst_->ResetIterator();
}

__host__ __device__
void ReadyList::Dev_Print() {
  printf("Ready List: ");
  for (auto crntInst = prirtyLst_->GetFrstElmnt(); crntInst != END;
       crntInst = prirtyLst_->GetNxtElmnt()) {
    printf(" %d", crntInst);
  }
  printf("\n");

  prirtyLst_->ResetIterator();
}

__host__ __device__
void ReadyList::AddLatestSubList_(ArrayList<InstCount> *lst) {
  assert(lst != NULL);

#ifdef IS_DEBUG_READY_LIST2
  Logger::GetLogStream() << "Adding to the ready list: ";
#endif

  // Start iterating from the bottom of the list to access the most recent
  // instructions first.
  SchedInstruction *crntInst;
  for (InstCount crntInstNum = lst->GetLastElmnt(); crntInstNum != END;
       crntInstNum = lst->GetPrevElmnt()) {
    // Once an instruction that is already in the ready list has been
    // encountered, this instruction and all the ones above it must be in the
    // ready list already.
    crntInst = dataDepGraph_->GetInstByIndx(crntInstNum);
    if (crntInst->IsInReadyList())
      break;
    AddInst(crntInst);
#ifdef IS_DEBUG_READY_LIST2
    Logger::GetLogStream() << crntInst->GetNum() << ", ";
#endif
    crntInst->PutInReadyList();
    latestSubLst_->InsrtElmnt(crntInstNum);
  }

#ifdef IS_DEBUG_READY_LIST2
  Logger::GetLogStream() << "\n";
#endif
}

__host__ __device__
void ReadyList::RemoveLatestSubList() {
#ifdef IS_DEBUG_READY_LIST2
  Logger::GetLogStream() << "Removing from the ready list: ";
#endif
  SchedInstruction *inst;
  for (InstCount instNum = latestSubLst_->GetFrstElmnt(); instNum != END;
       instNum = latestSubLst_->GetNxtElmnt()) {
    inst = dataDepGraph_->GetInstByIndx(instNum);
    assert(inst->IsInReadyList());
    inst->RemoveFromReadyList();
#ifdef IS_DEBUG_READY_LIST2
    Logger::GetLogStream() << inst->GetNum() << ", ";
#endif
  }

#ifdef IS_DEBUG_READY_LIST2
  Logger::GetLogStream() << "\n";
#endif
}

__host__ __device__
void ReadyList::ResetIterator() {
#ifdef __CUDA_ARCH__
  dev_prirtyLst_[GLOBALTID].ResetIterator();
#else
  prirtyLst_->ResetIterator(); 
#endif
}

__device__ __host__
void ReadyList::AddInst(SchedInstruction *inst) {
  bool changed;
  unsigned long key = CmputKey_(inst, false, changed);
  assert(changed == true);
#ifdef __CUDA_ARCH__
  dev_prirtyLst_[GLOBALTID].InsrtElmnt(inst->GetNum(), key, true);
#else
  prirtyLst_->InsrtElmnt(inst->GetNum(), key, true);
#endif
}

__device__ __host__
void ReadyList::AddList(ArrayList<InstCount> *lst) {
  SchedInstruction *crntInst;

  if (lst != NULL)
    for (InstCount crntInstNum = lst->GetFrstElmnt(); crntInstNum != END;
         crntInstNum = lst->GetNxtElmnt()) {
      crntInst = dataDepGraph_->GetInstByIndx(crntInstNum);
      AddInst(crntInst);
    }

  prirtyLst_->ResetIterator();
}

__host__ __device__
InstCount ReadyList::GetInstCnt() const {
#ifdef __CUDA_ARCH__
  return dev_prirtyLst_[GLOBALTID].GetElmntCnt();
#else
  return prirtyLst_->GetElmntCnt();
#endif
}

__host__ __device__
SchedInstruction *ReadyList::GetNextPriorityInst() {
#ifdef __CUDA_ARCH__
  return dataDepGraph_->GetInstByIndx(dev_prirtyLst_[GLOBALTID].GetNxtElmnt());
#else
  return dataDepGraph_->GetInstByIndx(prirtyLst_->GetNxtElmnt());
#endif
}

__host__ __device__
SchedInstruction *ReadyList::GetNextPriorityInst(unsigned long &key) {
#ifdef __CUDA_ARCH__
  int indx;
  SchedInstruction *inst = dataDepGraph_->
                    GetInstByIndx(dev_prirtyLst_[GLOBALTID].GetNxtElmnt(indx));
  key = dev_prirtyLst_[GLOBALTID].GetKey(indx);
  return inst;
#else
  int indx;
  SchedInstruction *inst = dataDepGraph_->
	            GetInstByIndx(prirtyLst_->GetNxtElmnt(indx));
  key = prirtyLst_->GetKey(indx);
  return inst;
#endif
}

__host__ __device__
void ReadyList::UpdatePriorities() {
  assert(prirts_.isDynmc);

  SchedInstruction *inst;
  bool instChanged = false;
  for (InstCount instNum = prirtyLst_->GetFrstElmnt(); instNum != END;
       instNum = prirtyLst_->GetNxtElmnt()) {
    inst = dataDepGraph_->GetInstByIndx(instNum);
    unsigned long key = CmputKey_(inst, true, instChanged);
    if (instChanged) {
      prirtyLst_->BoostElmnt(instNum, key);
    }
  }
}

__host__ __device__
void ReadyList::RemoveNextPriorityInst() {
#ifdef __CUDA_ARCH__
  dev_prirtyLst_[GLOBALTID].RmvCrntElmnt();
#else
  prirtyLst_->RmvCrntElmnt();
#endif
}

__host__ __device__
bool ReadyList::FindInst(SchedInstruction *inst, int &hitCnt) {
  return prirtyLst_->FindElmnt(inst->GetNum(), hitCnt);
}

__device__ __host__
void ReadyList::AddPrirtyToKey_(unsigned long &key, int16_t &keySize,
                                int16_t bitCnt, unsigned long val,
                                unsigned long maxVal) {
  assert(val <= maxVal);
  if (keySize > 0)
    key <<= bitCnt;
  key |= val;
  keySize += bitCnt;
}

__host__ __device__
unsigned long ReadyList::MaxPriority() { return maxPriority_; }

void ReadyList::AllocDevArraysForParallelACO(int numThreads) {
  size_t memSize;
  // Alloc dev array for dev_prirtyLst_
  memSize = sizeof(PriorityArrayList<InstCount>) * numThreads;
  gpuErrchk(hipMallocManaged(&dev_prirtyLst_, memSize));
}

void ReadyList::CopyPointersToDevice(ReadyList *dev_rdyLst, 
		                     DataDepGraph *dev_DDG, 
				     int numThreads) {
  size_t memSize;
  dev_rdyLst->dataDepGraph_ = dev_DDG;
  // Copy prirtyLst_
  memSize = sizeof(PriorityArrayList<InstCount>);
  for (int i = 0; i < numThreads; i++) {
    gpuErrchk(hipMemcpy(&dev_rdyLst->dev_prirtyLst_[i], prirtyLst_, memSize,
	  	         hipMemcpyHostToDevice));
  }
/*
  // debug
  Logger::Info("Testing prirtyLst_ copy to device:");
  for (int i = 0; i < numThreads; i++)
    Logger::Info("size of dev_rdyLst->dev_prirtyLst_[%d] = %d", i, dev_rdyLst->dev_prirtyLst_[i].size_);
*/
  // Alloc elmnts for each prirtyLst_ in one hipMalloc call
  InstCount *temp_arr;
  memSize = sizeof(InstCount) * prirtyLst_->maxSize_ * numThreads;
  gpuErrchk(hipMalloc(&temp_arr, memSize));
  // Assign a chunk of the large array to each prirtyLst_
  for (int i = 0; i < numThreads; i++)
    dev_rdyLst->dev_prirtyLst_[i].elmnts_ = &temp_arr[i * prirtyLst_->maxSize_];
  // Alloc keys for each prirtyLst_ in one hipMalloc call
  unsigned long *temp_ptr;
  memSize = sizeof(unsigned long) * prirtyLst_->maxSize_ * numThreads;
  gpuErrchk(hipMalloc(&temp_ptr, memSize));
  // Assign a chunk of the large array to each prirtyLst_
  for (int i = 0; i < numThreads; i++)
    dev_rdyLst->dev_prirtyLst_[i].keys_ = &temp_ptr[i * prirtyLst_->maxSize_];
/*
  memSize = sizeof(PriorityArrayList<InstCount>);
  gpuErrchk(hipMallocManaged(&dev_rdyLst->prirtyLst_, memSize));
  gpuErrchk(hipMemcpy(dev_rdyLst->prirtyLst_, prirtyLst_, memSize,
                       hipMemcpyHostToDevice));
  if (prirtyLst_->elmnts_) {
    memSize = sizeof(InstCount) * prirtyLst_->maxSize_;
    gpuErrchk(hipMalloc(&dev_rdyLst->prirtyLst_->elmnts_, memSize));
    memSize = sizeof(unsigned long) * prirtyLst_->maxSize_;
    gpuErrchk(hipMalloc(&dev_rdyLst->prirtyLst_->keys_, memSize));
  }
*/
}

void ReadyList::FreeDevicePointers(int numThreads) {
  hipFree(dev_prirtyLst_[0].keys_);
  hipFree(dev_prirtyLst_[0].elmnts_);
  hipFree(dev_prirtyLst_);
/*
  hipFree(prirtyLst_->elmnts_);
  hipFree(prirtyLst_->keys_);
  hipFree(prirtyLst_);
*/
}
